#define TOMO_OPS_EXPORTS
#include "tomo_dll.h"
#include "algorithm.h"

#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

hipError_t tomoFill(auto *a, size_t len, auto val, hipStream_t stream)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        thrust::fill(thrust::device.on(stream), a, a + len, val);
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFillF(float *a, size_t len, float val, hipStream_t stream)
{
    return tomoFill(a, len, val, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFillD(double *a, size_t len, double val, hipStream_t stream)
{
    return tomoFill(a, len, val, stream);
}

// hipError_t tomoGenerate(auto *a, size_t len, auto f, hipStream_t stream)
// {
//     using T = std::remove_cvref_t<decltype(*a)>;

//     if (len == 0)
//     {
//         return hipErrorInvalidValue;
//     }

//     try
//     {
//         thrust::generate(thrust::device.on(stream), a, a + len, f);
//     }
//     catch (const thrust::system_error &e)
//     {
//         if (e.code().category() == thrust::cuda_category())
//         {
//             return static_cast<hipError_t>(e.code().value());
//         }
//         else
//         {
//             return hipErrorUnknown;
//         }
//     }
//     catch (...)
//     {
//         return hipErrorUnknown;
//     }

//     return hipSuccess;
// }

hipError_t tomoSortDesc(auto *a, size_t len, hipStream_t stream)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        thrust::sort(thrust::device.on(stream), a, a + len, thrust::less<int>());
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortDescF(float *a, size_t len, hipStream_t stream)
{
    return tomoSortDesc(a, len, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortDescD(double *a, size_t len, hipStream_t stream)
{
    return tomoSortDesc(a, len, stream);
}

hipError_t tomoSortAsc(auto *a, size_t len, hipStream_t stream)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        thrust::sort(thrust::device.on(stream), a, a + len, thrust::greater<int>());
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortAscF(float *a, size_t len, hipStream_t stream)
{
    return tomoSortAsc(a, len, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortAscD(double *a, size_t len, hipStream_t stream)
{
    return tomoSortAsc(a, len, stream);
}

hipError_t tomoFind(auto *a, size_t len, auto val, hipStream_t stream, size_t *i)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        auto it = thrust::find(thrust::device.on(stream), a, a + len, val);
        *i = std::distance(a, it);
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFindF(float *a, size_t len, float val, hipStream_t stream, size_t *i)
{
    return tomoFind(a, len, val, stream, i);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFindD(double *a, size_t len, double val, hipStream_t stream, size_t *i)
{
    return tomoFind(a, len, val, stream, i);
}
