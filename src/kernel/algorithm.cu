#include "hip/hip_runtime.h"
#define TOMO_OPS_EXPORTS
#include "tomo_dll.h"
#include "algorithm.h"

#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

hipError_t tomoFill(auto *a, size_t len, auto val, hipStream_t stream)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        thrust::fill(thrust::cuda::par_nosync.on(stream), a, a + len, val);
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFillH(__half_raw *a, size_t len, __half_raw val, hipStream_t stream)
{
    return tomoFill((__half*)a, len,(__half) val, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFillB(__hip_bfloat16_raw *a, size_t len, __hip_bfloat16_raw val, hipStream_t stream)
{
    return tomoFill((__hip_bfloat16*)a, len, (__hip_bfloat16)val, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFillF(float *a, size_t len, float val, hipStream_t stream)
{
    return tomoFill(a, len, val, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFillD(double *a, size_t len, double val, hipStream_t stream)
{
    return tomoFill(a, len, val, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFillUZ(size_t *a, size_t len, size_t val, hipStream_t stream)
{
    return tomoFill(a, len, val, stream);
}

hipError_t tomoSortDesc(auto *a, size_t len, hipStream_t stream)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        thrust::sort(thrust::device.on(stream), a, a + len, thrust::less<T>());
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortDescH(__half_raw *a, size_t len, hipStream_t stream)
{
    return tomoSortDesc((__half*)a, len, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortDescB(__hip_bfloat16_raw *a, size_t len, hipStream_t stream)
{
    return tomoSortDesc((__hip_bfloat16*)a, len, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortDescF(float *a, size_t len, hipStream_t stream)
{
    return tomoSortDesc(a, len, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortDescD(double *a, size_t len, hipStream_t stream)
{
    return tomoSortDesc(a, len, stream);
}

hipError_t tomoSortAsc(auto *a, size_t len, hipStream_t stream)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        thrust::sort(thrust::device.on(stream), a, a + len, thrust::greater<T>());
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortAscH(__half_raw *a, size_t len, hipStream_t stream)
{
    return tomoSortAsc((__half*)a, len, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortAscB(__hip_bfloat16_raw *a, size_t len, hipStream_t stream)
{
    return tomoSortAsc((__hip_bfloat16*)a, len, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortAscF(float *a, size_t len, hipStream_t stream)
{
    return tomoSortAsc(a, len, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSortAscD(double *a, size_t len, hipStream_t stream)
{
    return tomoSortAsc(a, len, stream);
}

hipError_t tomoFind(auto *a, size_t len, auto val, hipStream_t stream, size_t *i)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        auto it = thrust::find(thrust::device.on(stream), a, a + len, val);
        *i = std::distance(a, it);
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFindH(__half_raw *a, size_t len, __half_raw val, hipStream_t stream, size_t *i)
{
    return tomoFind((__half*)a, len, (__half)val, stream, i);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFindB(__hip_bfloat16_raw *a, size_t len, __hip_bfloat16_raw val, hipStream_t stream, size_t *i)
{
    return tomoFind((__hip_bfloat16*)a, len, (__hip_bfloat16)val, stream, i);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFindF(float *a, size_t len, float val, hipStream_t stream, size_t *i)
{
    return tomoFind(a, len, val, stream, i);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoFindD(double *a, size_t len, double val, hipStream_t stream, size_t *i)
{
    return tomoFind(a, len, val, stream, i);
}
