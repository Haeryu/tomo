#include "hip/hip_runtime.h"
#define TOMO_OPS_EXPORTS
#include "tomo_dll.h"
#include "batch.h"

#include "float_op.cuh"

template <typename T>
__global__ void tomoGetItemKernel(
    T const *x, T *y,
    size_t const *in_shape, size_t const *out_shape,
    size_t const *in_strides, size_t const *out_strides,
    size_t const *starts, size_t const *steps,
    size_t nd, size_t out_size)
{
    size_t out_idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
    {
        return;
    }

    // Unravel out_idx to out_coords
    size_t out_coords[32]; // max_rank = 4
    size_t tmp = out_idx;
    for (ptrdiff_t d = (ptrdiff_t)nd - 1; d >= 0; --d)
    {
        out_coords[d] = tmp % out_shape[d];
        tmp /= out_shape[d];
    }

    // Compute corresponding input coordinates
    size_t in_coords[32];
    for (size_t d = 0; d < nd; ++d)
    {
        in_coords[d] = starts[d] + out_coords[d] * steps[d];
    }

    // Compute input index
    size_t in_idx = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        in_idx += in_coords[d] * in_strides[d];
    }

    // Copy value
    y[out_idx] = x[in_idx];
}

template <typename T>
__global__ void tomoGetItemGradKernel(
    T const *gy, T *gx,
    size_t const *in_shape, size_t const *out_shape,
    size_t const *in_strides, size_t const *out_strides,
    size_t const *starts, size_t const *steps,
    size_t nd, size_t out_size)
{
    size_t out_idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
    {
        return;
    }

    // Unravel out_idx to out_coords
    size_t out_coords[32];
    size_t tmp = out_idx;
    for (ptrdiff_t d = (ptrdiff_t)nd - 1; d >= 0; --d)
    {
        out_coords[d] = tmp % out_shape[d];
        tmp /= out_shape[d];
    }

    // Compute corresponding input coordinates
    size_t in_coords[32];
    for (size_t d = 0; d < nd; ++d)
    {
        in_coords[d] = starts[d] + out_coords[d] * steps[d];
    }

    // Compute input index
    size_t in_idx = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        in_idx += in_coords[d] * in_strides[d];
    }

    // Add gradient (no overlap in standard slicing)
    // gx[in_idx] = gx[in_idx] + gy[out_idx];
    if constexpr (std::is_same_v<T, __half_raw>)
    {
        atomicAdd(reinterpret_cast<__half*>(&gx[in_idx]),  static_cast<__half>(gy[out_idx]));
    }
    else if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
    {

        atomicAdd(reinterpret_cast<__hip_bfloat16*>(&gx[in_idx]),  static_cast<__hip_bfloat16>(gy[out_idx]));
    }
    else
    {

        atomicAdd(&gx[in_idx], gy[out_idx]);
    }
}

template <typename T>
hipError_t tomoGetItem(
    T const *x, T *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{

    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides, *d_starts, *d_steps;
    hipError_t err;

    err = hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_starts, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_steps, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }

    err = hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_starts, starts, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_steps, steps, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }

    int const threads = 256;
    int const blocks = (int)(out_size + threads - 1) / threads;
    tomoGetItemKernel<T><<<blocks, threads, 0, stream>>>(
        x, y, d_in_shape, d_out_shape, d_in_strides, d_out_strides, d_starts, d_steps, nd, out_size);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        return err;
    }

    hipFreeAsync(d_in_shape, stream);
    hipFreeAsync(d_out_shape, stream);
    hipFreeAsync(d_in_strides, stream);
    hipFreeAsync(d_out_strides, stream);
    hipFreeAsync(d_starts, stream);
    hipFreeAsync(d_steps, stream);

    return hipSuccess;
}

template <typename T>
__global__ void tomoSetItemKernel(
    T const *src, T *dest,
    size_t const *src_shape, size_t const *dest_shape,
    size_t const *src_strides, size_t const *dest_strides,
    size_t const *starts, size_t const *steps,
    size_t nd, size_t src_size)
{
    // Get the global thread index
    size_t src_idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (src_idx >= src_size) {
        return;
    }

    // Unravel src_idx into source coordinates
    size_t src_coords[32]; // Assuming max_rank is 32
    size_t tmp = src_idx;
    for (ptrdiff_t d = (ptrdiff_t)nd - 1; d >= 0; --d) {
        src_coords[d] = tmp % src_shape[d];
        tmp /= src_shape[d];
    }

    // Compute corresponding destination coordinates using starts and steps
    size_t dest_coords[32];
    for (size_t d = 0; d < nd; ++d) {
        dest_coords[d] = starts[d] + src_coords[d] * steps[d];
    }

    // Compute destination index using strides
    size_t dest_idx = 0;
    for (size_t d = 0; d < nd; ++d) {
        dest_idx += dest_coords[d] * dest_strides[d];
    }

    // Copy value from source to destination
    dest[dest_idx] = src[src_idx];
}

template <typename T>
hipError_t tomoSetItem(
    T const *src, T *dest,
    size_t const *src_shape, size_t src_shape_len,
    size_t const *dest_shape, size_t dest_shape_len,
    size_t const *src_strides, size_t src_strides_len,
    size_t const *dest_strides, size_t dest_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t src_size,
    hipStream_t stream)
{
    // Device memory pointers
    size_t *d_src_shape, *d_dest_shape, *d_src_strides, *d_dest_strides, *d_starts, *d_steps;
    hipError_t err;

    // Allocate device memory
    err = hipMallocAsync(&d_src_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess) return err;
    err = hipMallocAsync(&d_dest_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess) return err;
    err = hipMallocAsync(&d_src_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess) return err;
    err = hipMallocAsync(&d_dest_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess) return err;
    err = hipMallocAsync(&d_starts, nd * sizeof(size_t), stream);
    if (err != hipSuccess) return err;
    err = hipMallocAsync(&d_steps, nd * sizeof(size_t), stream);
    if (err != hipSuccess) return err;

    // Copy data to device
    err = hipMemcpyAsync(d_src_shape, src_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) return err;
    err = hipMemcpyAsync(d_dest_shape, dest_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) return err;
    err = hipMemcpyAsync(d_src_strides, src_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) return err;
    err = hipMemcpyAsync(d_dest_strides, dest_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) return err;
    err = hipMemcpyAsync(d_starts, starts, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) return err;
    err = hipMemcpyAsync(d_steps, steps, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) return err;

    // Launch the kernel
    int const threads = 256;
    int const blocks = (int)(src_size + threads - 1) / threads;
    tomoSetItemKernel<T><<<blocks, threads, 0, stream>>>(
        src, dest, d_src_shape, d_dest_shape, d_src_strides, d_dest_strides, d_starts, d_steps, nd, src_size);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) return err;

    // Free device memory
    hipFreeAsync(d_src_shape, stream);
    hipFreeAsync(d_dest_shape, stream);
    hipFreeAsync(d_src_strides, stream);
    hipFreeAsync(d_dest_strides, stream);
    hipFreeAsync(d_starts, stream);
    hipFreeAsync(d_steps, stream);

    return hipSuccess;
}

template <typename T>
hipError_t tomoGetItemGrad(
    T const *gy, T *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{

    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides, *d_starts, *d_steps;
    hipError_t err;

    err = hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_starts, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_steps, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }

    err = hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_starts, starts, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_steps, steps, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }

    int const threads = 256;
    int const blocks = (int)(out_size + threads - 1) / threads;
    tomoGetItemGradKernel<T><<<blocks, threads, 0, stream>>>(
        gy, gx, d_in_shape, d_out_shape, d_in_strides, d_out_strides, d_starts, d_steps, nd, out_size);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        return err;
    }

    hipFreeAsync(d_in_shape, stream);
    hipFreeAsync(d_out_shape, stream);
    hipFreeAsync(d_in_strides, stream);
    hipFreeAsync(d_out_strides, stream);
    hipFreeAsync(d_starts, stream);
    hipFreeAsync(d_steps, stream);

    return hipSuccess;
}

template <typename T>
__global__ void tomoOneHotKernel(
    size_t const *indices, // Input: 1D array of class indices [batch_size]
    T *one_hot,            // Output: 2D one-hot tensor [batch_size, num_classes]
    size_t batch_size,
    size_t num_classes)
{
    size_t idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size)
        return;

    // Get the class index for this batch item
    size_t class_idx = indices[idx];
    if (class_idx >= num_classes)
        return; // Safety check

    // Set the corresponding element to 1
    one_hot[idx * num_classes + class_idx] = static_cast<T>(1.0);
}

template <typename T>
hipError_t tomoOneHot(
    size_t const *indices,
    T *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    // Zero-initialize the output tensor
    hipError_t err = hipMemsetAsync(one_hot, 0, batch_size * num_classes * sizeof(T), stream);
    if (err != hipSuccess)
        return err;

    // Launch the kernel
    int const threads = 256;
    int const blocks = (int)(batch_size + threads - 1) / threads;
    tomoOneHotKernel<T><<<blocks, threads, 0, stream>>>(
        indices, one_hot, batch_size, num_classes);

    err = hipGetLastError();
    if (err != hipSuccess)
        return err;

    return hipSuccess;
}

// ----- GetItem Wrappers -----
// Half
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemH(
    __half_raw const *x,
    __half_raw *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItem<__half_raw>(
        x, y,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemB(
    __hip_bfloat16_raw const *x,
    __hip_bfloat16_raw *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItem<__hip_bfloat16_raw>(
        x, y,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemF(
    float const *x,
    float *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItem<float>(
        x, y,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemD(
    double const *x,
    double *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItem<double>(
        x, y,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Half
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSetItemH(
    __half_raw const *src,
    __half_raw *dest,
    size_t const *src_shape, size_t src_shape_len,
    size_t const *dest_shape, size_t dest_shape_len,
    size_t const *src_strides, size_t src_strides_len,
    size_t const *dest_strides, size_t dest_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t src_size,
    hipStream_t stream)
{
    return tomoSetItem<__half_raw>(
        src, dest,
        src_shape, src_shape_len,
        dest_shape, dest_shape_len,
        src_strides, src_strides_len,
        dest_strides, dest_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, src_size,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSetItemB(
    __hip_bfloat16_raw const *src,
    __hip_bfloat16_raw *dest,
    size_t const *src_shape, size_t src_shape_len,
    size_t const *dest_shape, size_t dest_shape_len,
    size_t const *src_strides, size_t src_strides_len,
    size_t const *dest_strides, size_t dest_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t src_size,
    hipStream_t stream)
{
    return tomoSetItem<__hip_bfloat16_raw>(
        src, dest,
        src_shape, src_shape_len,
        dest_shape, dest_shape_len,
        src_strides, src_strides_len,
        dest_strides, dest_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, src_size,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSetItemF(
    float const *src,
    float *dest,
    size_t const *src_shape, size_t src_shape_len,
    size_t const *dest_shape, size_t dest_shape_len,
    size_t const *src_strides, size_t src_strides_len,
    size_t const *dest_strides, size_t dest_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t src_size,
    hipStream_t stream)
{
    return tomoSetItem<float>(
        src, dest,
        src_shape, src_shape_len,
        dest_shape, dest_shape_len,
        src_strides, src_strides_len,
        dest_strides, dest_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, src_size,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSetItemD(
    double const *src,
    double *dest,
    size_t const *src_shape, size_t src_shape_len,
    size_t const *dest_shape, size_t dest_shape_len,
    size_t const *src_strides, size_t src_strides_len,
    size_t const *dest_strides, size_t dest_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t src_size,
    hipStream_t stream)
{
    return tomoSetItem<double>(
        src, dest,
        src_shape, src_shape_len,
        dest_shape, dest_shape_len,
        src_strides, src_strides_len,
        dest_strides, dest_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, src_size,
        stream);
}


// ----- GetItemGrad Wrappers -----
// Half
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemGradH(
    __half_raw const *gy,
    __half_raw *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItemGrad<__half_raw>(
        gy, gx,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemGradB(
    __hip_bfloat16_raw const *gy,
    __hip_bfloat16_raw *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItemGrad<__hip_bfloat16_raw>(
        gy, gx,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemGradF(
    float const *gy,
    float *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItemGrad<float>(
        gy, gx,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemGradD(
    double const *gy,
    double *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItemGrad<double>(
        gy, gx,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Wrappers for specific types
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoOneHotH(
    size_t const *indices,
    __half_raw *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    return tomoOneHot<__half_raw>(indices, one_hot, batch_size, num_classes, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoOneHotB(
    size_t const *indices,
    __hip_bfloat16_raw *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    return tomoOneHot<__hip_bfloat16_raw>(indices, one_hot, batch_size, num_classes, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoOneHotF(
    size_t const *indices,
    float *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    return tomoOneHot<float>(indices, one_hot, batch_size, num_classes, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoOneHotD(
    size_t const *indices,
    double *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    return tomoOneHot<double>(indices, one_hot, batch_size, num_classes, stream);
}