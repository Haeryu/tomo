#include "hip/hip_runtime.h"
#define TOMO_OPS_EXPORTS
#include "tomo_dll.h"
#include "batch.h"

#include "float_op.cuh"

template <typename T>
__global__ void tomoGetItemKernel(
    T const *x, T *y,
    size_t const *in_shape, size_t const *out_shape,
    size_t const *in_strides, size_t const *out_strides,
    size_t const *starts, size_t const *steps,
    size_t nd, size_t out_size)
{
    size_t out_idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
    {
        return;
    }

    // Unravel out_idx to out_coords
    size_t out_coords[32]; // max_rank = 4
    size_t tmp = out_idx;
    for (ptrdiff_t d = (ptrdiff_t)nd - 1; d >= 0; --d)
    {
        out_coords[d] = tmp % out_shape[d];
        tmp /= out_shape[d];
    }

    // Compute corresponding input coordinates
    size_t in_coords[32];
    for (size_t d = 0; d < nd; ++d)
    {
        in_coords[d] = starts[d] + out_coords[d] * steps[d];
    }

    // Compute input index
    size_t in_idx = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        in_idx += in_coords[d] * in_strides[d];
    }

    // Copy value
    y[out_idx] = x[in_idx];
}

template <typename T>
__global__ void tomoGetItemGradKernel(
    T const *gy, T *gx,
    size_t const *in_shape, size_t const *out_shape,
    size_t const *in_strides, size_t const *out_strides,
    size_t const *starts, size_t const *steps,
    size_t nd, size_t out_size)
{
    size_t out_idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
    {
        return;
    }

    // Unravel out_idx to out_coords
    size_t out_coords[32];
    size_t tmp = out_idx;
    for (ptrdiff_t d = (ptrdiff_t)nd - 1; d >= 0; --d)
    {
        out_coords[d] = tmp % out_shape[d];
        tmp /= out_shape[d];
    }

    // Compute corresponding input coordinates
    size_t in_coords[32];
    for (size_t d = 0; d < nd; ++d)
    {
        in_coords[d] = starts[d] + out_coords[d] * steps[d];
    }

    // Compute input index
    size_t in_idx = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        in_idx += in_coords[d] * in_strides[d];
    }

    // Add gradient (no overlap in standard slicing)
    // gx[in_idx] = gx[in_idx] + gy[out_idx];
    if constexpr (std::is_same_v<T, __half_raw>)
    {
        atomicAdd(reinterpret_cast<__half*>(&gx[in_idx]),  static_cast<__half>(gy[out_idx]));
    }
    else if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
    {

        atomicAdd(reinterpret_cast<__hip_bfloat16*>(&gx[in_idx]),  static_cast<__hip_bfloat16>(gy[out_idx]));
    }
    else
    {

        atomicAdd(&gx[in_idx], gy[out_idx]);
    }
}

template <typename T>
hipError_t tomoGetItem(
    T const *x, T *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{

    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides, *d_starts, *d_steps;
    hipError_t err;

    err = hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_starts, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_steps, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }

    err = hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_starts, starts, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_steps, steps, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }

    int const threads = 256;
    int const blocks = (int)(out_size + threads - 1) / threads;
    tomoGetItemKernel<T><<<blocks, threads, 0, stream>>>(
        x, y, d_in_shape, d_out_shape, d_in_strides, d_out_strides, d_starts, d_steps, nd, out_size);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        return err;
    }

    hipFreeAsync(d_in_shape, stream);
    hipFreeAsync(d_out_shape, stream);
    hipFreeAsync(d_in_strides, stream);
    hipFreeAsync(d_out_strides, stream);
    hipFreeAsync(d_starts, stream);
    hipFreeAsync(d_steps, stream);

    return hipSuccess;
}

template <typename T>
hipError_t tomoGetItemGrad(
    T const *gy, T *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{

    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides, *d_starts, *d_steps;
    hipError_t err;

    err = hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_starts, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMallocAsync(&d_steps, nd * sizeof(size_t), stream);
    if (err != hipSuccess)
    {
        return err;
    }

    err = hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_starts, starts, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }
    err = hipMemcpyAsync(d_steps, steps, nd * sizeof(size_t), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        return err;
    }

    int const threads = 256;
    int const blocks = (int)(out_size + threads - 1) / threads;
    tomoGetItemGradKernel<T><<<blocks, threads, 0, stream>>>(
        gy, gx, d_in_shape, d_out_shape, d_in_strides, d_out_strides, d_starts, d_steps, nd, out_size);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        return err;
    }

    hipFreeAsync(d_in_shape, stream);
    hipFreeAsync(d_out_shape, stream);
    hipFreeAsync(d_in_strides, stream);
    hipFreeAsync(d_out_strides, stream);
    hipFreeAsync(d_starts, stream);
    hipFreeAsync(d_steps, stream);

    return hipSuccess;
}

template <typename T>
__global__ void tomoOneHotKernel(
    size_t const *indices, // Input: 1D array of class indices [batch_size]
    T *one_hot,            // Output: 2D one-hot tensor [batch_size, num_classes]
    size_t batch_size,
    size_t num_classes)
{
    size_t idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size)
        return;

    // Get the class index for this batch item
    size_t class_idx = indices[idx];
    if (class_idx >= num_classes)
        return; // Safety check

    // Set the corresponding element to 1
    one_hot[idx * num_classes + class_idx] = static_cast<T>(1.0);
}

template <typename T>
hipError_t tomoOneHot(
    size_t const *indices,
    T *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    // Zero-initialize the output tensor
    hipError_t err = hipMemsetAsync(one_hot, 0, batch_size * num_classes * sizeof(T), stream);
    if (err != hipSuccess)
        return err;

    // Launch the kernel
    int const threads = 256;
    int const blocks = (int)(batch_size + threads - 1) / threads;
    tomoOneHotKernel<T><<<blocks, threads, 0, stream>>>(
        indices, one_hot, batch_size, num_classes);

    err = hipGetLastError();
    if (err != hipSuccess)
        return err;

    return hipSuccess;
}

// ----- GetItem Wrappers -----
// Half
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemH(
    __half_raw const *x,
    __half_raw *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItem<__half_raw>(
        x, y,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemB(
    __hip_bfloat16_raw const *x,
    __hip_bfloat16_raw *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItem<__hip_bfloat16_raw>(
        x, y,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemF(
    float const *x,
    float *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItem<float>(
        x, y,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemD(
    double const *x,
    double *y,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItem<double>(
        x, y,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// ----- GetItemGrad Wrappers -----
// Half
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemGradH(
    __half_raw const *gy,
    __half_raw *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItemGrad<__half_raw>(
        gy, gx,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemGradB(
    __hip_bfloat16_raw const *gy,
    __hip_bfloat16_raw *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItemGrad<__hip_bfloat16_raw>(
        gy, gx,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemGradF(
    float const *gy,
    float *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItemGrad<float>(
        gy, gx,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoGetItemGradD(
    double const *gy,
    double *gx,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *starts, size_t starts_len,
    size_t const *steps, size_t steps_len,
    size_t nd, size_t out_size,
    hipStream_t stream)
{
    return tomoGetItemGrad<double>(
        gy, gx,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        starts, starts_len,
        steps, steps_len,
        nd, out_size,
        stream);
}

// Wrappers for specific types
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoOneHotH(
    size_t const *indices,
    __half_raw *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    return tomoOneHot<__half_raw>(indices, one_hot, batch_size, num_classes, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoOneHotB(
    size_t const *indices,
    __hip_bfloat16_raw *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    return tomoOneHot<__hip_bfloat16_raw>(indices, one_hot, batch_size, num_classes, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoOneHotF(
    size_t const *indices,
    float *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    return tomoOneHot<float>(indices, one_hot, batch_size, num_classes, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoOneHotD(
    size_t const *indices,
    double *one_hot,
    size_t batch_size,
    size_t num_classes,
    hipStream_t stream)
{
    return tomoOneHot<double>(indices, one_hot, batch_size, num_classes, stream);
}