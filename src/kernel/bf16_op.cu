#include "hip/hip_runtime.h"
#define TOMO_OPS_EXPORTS
#include "tomo_dll.h"
#include "bf16_op.h"

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "float_op.cuh"

TOMO_EXTERN_C TOMO_OPS_API __hip_bfloat16_raw tomoBf16Add(__hip_bfloat16_raw lh, __hip_bfloat16_raw rh)
{
    return __hip_bfloat16{lh} + __hip_bfloat16{rh};
}
TOMO_EXTERN_C TOMO_OPS_API __hip_bfloat16_raw tomoBf16Neg(__hip_bfloat16_raw lh)
{
    return -__hip_bfloat16{lh};
}
TOMO_EXTERN_C TOMO_OPS_API __hip_bfloat16_raw tomoBf16Sub(__hip_bfloat16_raw lh, __hip_bfloat16_raw rh)
{
    return __hip_bfloat16{lh} - __hip_bfloat16{rh};
}
TOMO_EXTERN_C TOMO_OPS_API __hip_bfloat16_raw tomoBf16Mul(__hip_bfloat16_raw lh, __hip_bfloat16_raw rh)
{
    return __hip_bfloat16{lh} * __hip_bfloat16{rh};
}
TOMO_EXTERN_C TOMO_OPS_API __hip_bfloat16_raw tomoBf16Div(__hip_bfloat16_raw lh, __hip_bfloat16_raw rh)
{
    return __hip_bfloat16{lh} / __hip_bfloat16{rh};
}