#include "hip/hip_runtime.h"
#define TOMO_OPS_EXPORTS
#include "tomo_dll.h"

#include "tensor_ops.h"

#include <limits>
#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>

#include "float_op.cuh"

#define MAX_ND 32

#define CHECK_CUDA(call)        \
    do                          \
    {                           \
        hipError_t err = call; \
        if (err != hipSuccess) \
            return err;         \
    } while (0)

template <typename T>
hipError_t tomoBroadcastTo(
    const T *d_in, T *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t in_size, size_t out_size, size_t nd,
    hipStream_t stream)
{
    // Validate inputs

    // Device buffers (could be cached in a real system)
    size_t *d_in_shape, *d_out_shape, *d_in_stride;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_stride, nd * sizeof(size_t), stream));

    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_stride, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));

    auto first = thrust::counting_iterator<size_t>{0};
    auto last = thrust::counting_iterator<size_t>{out_size};

    thrust::transform(thrust::cuda::par_nosync.on(stream),
                      thrust::counting_iterator<size_t>{0},
                      thrust::counting_iterator<size_t>{out_size},
                      thrust::device_pointer_cast(d_out),
                      [=] __device__(size_t out_index)
                      {
                          size_t coords[MAX_ND];
                          size_t tmp = out_index;
                          for (ptrdiff_t d = nd - 1; d >= 0; --d)
                          {
                              size_t dim_size = d_out_shape[d];
                              coords[d] = tmp % dim_size;
                              tmp /= dim_size;
                          }
                          size_t in_index = 0;
                          for (size_t d = 0; d < nd; ++d)
                          {
                              size_t c;
                              if (d_in_shape[d] == 1)
                              {
                                  c = 0; // Broadcasting
                              }
                              else if (d_out_shape[d] % d_in_shape[d] == 0)
                              {
                                  c = coords[d] % d_in_shape[d]; // Tiling
                              }
                              else
                              {
                                  c = coords[d]; // Direct mapping (if shapes match)
                              }
                              in_index = in_index + c * d_in_stride[d];
                          }
                          return d_in[in_index];
                      });

    CHECK_CUDA(hipFreeAsync(d_in_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_out_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_in_stride, stream));
    return hipSuccess;
}

// // Custom reduction kernel for tomoSumTo
// template <typename T>
// __global__ void tomoSumToKernel(
//     const T *d_in, T *d_out,
//     const size_t *d_in_shape, const size_t *d_out_shape,
//     const size_t *d_in_strides, const size_t *d_out_strides,
//     size_t in_size, size_t out_size, size_t nd)
// {
//     size_t out_idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (out_idx >= out_size)
//         return;

//     // Unravel out_idx to out_coords
//     size_t out_coords[MAX_ND];
//     size_t tmp = out_idx;
//     for (ptrdiff_t d = nd - 1; d >= 0; --d)
//     {
//         size_t dim_size = d_out_shape[d];
//         out_coords[d] = tmp % dim_size;
//         tmp /= dim_size;
//     }

//     // Compute sum over input elements mapping to this output index
//     T sum = T(0);
//     size_t in_coords[MAX_ND];
//     for (size_t in_idx = 0; in_idx < in_size; ++in_idx)
//     {
//         size_t unravel = in_idx;
//         bool matches = true;
//         for (ptrdiff_t d = nd - 1; d >= 0; --d)
//         {
//             size_t dim_size = d_in_shape[d];
//             in_coords[d] = unravel % dim_size;
//             unravel /= dim_size;
//             if (d_out_shape[d] != 1)
//             {
//                 size_t out_c = out_coords[d];
//                 size_t in_c = (d_in_shape[d] == 1) ? 0 : in_coords[d];
//                 if (out_c != in_c)
//                 {
//                     matches = false;
//                     break;
//                 }
//             }
//         }
//         if (matches)
//         {
//             size_t in_offset = 0;
//             for (size_t d = 0; d < nd; ++d)
//             {
//                 in_offset = in_offset + in_coords[d] * d_in_strides[d];
//             }
//             sum = sum + d_in[in_offset];
//         }
//     }
//     d_out[out_idx] = sum;
// }

// template <typename T>
// __global__ void tomoSumToKernel(
//     const T* d_in,            // Input tensor data
//     const size_t* in_shape,   // Input tensor shape
//     const size_t* in_strides, // Input tensor strides
//     T* d_out,                 // Output tensor data
//     const size_t* out_shape,  // Output tensor shape
//     const size_t* out_strides,// Output tensor strides
//     size_t out_size,          // Total number of output elements
//     size_t nd                 // Number of dimensions
// ) {
//     // Maximum supported dimensions
//     //const size_t MAX_DIMS = 10;

//     // Each block handles one output element
//     size_t out_idx = blockIdx.x;
//     if (out_idx >= out_size) return;

//     // Compute output coordinates from out_idx
//     size_t out_coords[MAX_ND];
//     size_t tmp = out_idx;
//     for (ptrdiff_t d = nd - 1; d >= 0; --d) {
//         out_coords[d] = tmp % out_shape[d];
//         tmp /= out_shape[d];
//     }

//     // Compute output offset using out_strides
//     size_t out_offset = 0;
//     for (size_t d = 0; d < nd; ++d) {
//         out_offset += out_coords[d] * out_strides[d];
//     }

//     // Compute base offset in the input tensor
//     size_t base_offset = 0;
//     for (size_t d = 0; d < nd; ++d) {
//         size_t in_c = (out_shape[d] == 1) ? 0 : out_coords[d];
//         base_offset += in_c * in_strides[d];
//     }

//     // Identify reduced dimensions
//     size_t reduced_dims[MAX_ND];
//     size_t reduced_sizes[MAX_ND];
//     size_t num_reduced = 0;
//     for (size_t d = 0; d < nd; ++d) {
//         if (out_shape[d] == 1 && in_shape[d] > 1) {
//             reduced_dims[num_reduced] = d;
//             reduced_sizes[num_reduced] = in_shape[d];
//             num_reduced++;
//         }
//     }

//     // Compute total number of elements to sum
//     size_t N = 1;
//     for (size_t j = 0; j < num_reduced; ++j) {
//         N *= reduced_sizes[j];
//     }

//     // Each thread computes a partial sum
//     T partial_sum = (T)0.0;
//     for (size_t i = threadIdx.x; i < N; i += blockDim.x) {
//         // Compute coordinates in the reduced dimensions from flattened index i
//         size_t reduced_coords[MAX_ND];
//         size_t tmp_i = i;
//         for (ptrdiff_t j = num_reduced - 1; j >= 0; --j) {
//             reduced_coords[j] = tmp_i % reduced_sizes[j];
//             tmp_i /= reduced_sizes[j];
//         }

//         // Compute input offset
//         size_t offset = base_offset;
//         for (size_t j = 0; j < num_reduced; ++j) {
//             size_t d = reduced_dims[j];
//             offset += reduced_coords[j] * in_strides[d];
//         }

//         // Accumulate into partial sum
//         partial_sum = partial_sum + d_in[offset];
//     }

//     // Use shared memory for reduction within the block
//     extern __shared__ T shared_sums[];
//     shared_sums[threadIdx.x] = partial_sum;
//     __syncthreads();

//     // Perform parallel reduction in shared memory
//     // Assumes blockDim.x is a power of two
//     for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
//         if (threadIdx.x < s) {
//             shared_sums[threadIdx.x] = shared_sums[threadIdx.x] + shared_sums[threadIdx.x + s];
//         }
//         __syncthreads();
//     }

//     // Thread 0 writes the final sum to the output
//     if (threadIdx.x == 0) {
//         d_out[out_offset] = shared_sums[0];
//     }
// }

__global__ void tomoSumToKernel(
    const __half_raw *d_in,    // Input tensor data
    const size_t *in_shape,    // Input tensor shape
    const size_t *in_strides,  // Input tensor strides
    __half_raw *d_out,         // Output tensor data
    const size_t *out_shape,   // Output tensor shape
    const size_t *out_strides, // Output tensor strides
    size_t out_size,           // Total number of output elements
    size_t nd                  // Number of dimensions
)
{
    // Maximum supported dimensions
    // const size_t MAX_DIMS = 10;

    // Each block handles one output element
    size_t out_idx = blockIdx.x;
    if (out_idx >= out_size)
        return;

    // Compute output coordinates from out_idx
    size_t out_coords[MAX_ND];
    size_t tmp = out_idx;
    for (ptrdiff_t d = nd - 1; d >= 0; --d)
    {
        out_coords[d] = tmp % out_shape[d];
        tmp /= out_shape[d];
    }

    // Compute output offset using out_strides
    size_t out_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        out_offset += out_coords[d] * out_strides[d];
    }

    // Compute base offset in the input tensor
    size_t base_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        size_t in_c = (out_shape[d] == 1) ? 0 : out_coords[d];
        base_offset += in_c * in_strides[d];
    }

    // Identify reduced dimensions
    size_t reduced_dims[MAX_ND];
    size_t reduced_sizes[MAX_ND];
    size_t num_reduced = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        if (out_shape[d] == 1 && in_shape[d] > 1)
        {
            reduced_dims[num_reduced] = d;
            reduced_sizes[num_reduced] = in_shape[d];
            num_reduced++;
        }
    }

    // Compute total number of elements to sum
    size_t N = 1;
    for (size_t j = 0; j < num_reduced; ++j)
    {
        N *= reduced_sizes[j];
    }

    // Each thread computes a partial sum
    __half_raw partial_sum = (__half_raw)0.0;
    for (size_t i = threadIdx.x; i < N; i += blockDim.x)
    {
        // Compute coordinates in the reduced dimensions from flattened index i
        size_t reduced_coords[MAX_ND];
        size_t tmp_i = i;
        for (ptrdiff_t j = num_reduced - 1; j >= 0; --j)
        {
            reduced_coords[j] = tmp_i % reduced_sizes[j];
            tmp_i /= reduced_sizes[j];
        }

        // Compute input offset
        size_t offset = base_offset;
        for (size_t j = 0; j < num_reduced; ++j)
        {
            size_t d = reduced_dims[j];
            offset += reduced_coords[j] * in_strides[d];
        }

        // Accumulate into partial sum
        partial_sum = partial_sum + d_in[offset];
    }

    // Use shared memory for reduction within the block
    extern __shared__ __half_raw shared_sumsh[];
    shared_sumsh[threadIdx.x] = partial_sum;
    __syncthreads();

    // Perform parallel reduction in shared memory
    // Assumes blockDim.x is a power of two
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            shared_sumsh[threadIdx.x] = shared_sumsh[threadIdx.x] + shared_sumsh[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Thread 0 writes the final sum to the output
    if (threadIdx.x == 0)
    {
        d_out[out_offset] = shared_sumsh[0];
    }
}

__global__ void tomoSumToKernel(
    const __hip_bfloat16_raw *d_in, // Input tensor data
    const size_t *in_shape,        // Input tensor shape
    const size_t *in_strides,      // Input tensor strides
    __hip_bfloat16_raw *d_out,      // Output tensor data
    const size_t *out_shape,       // Output tensor shape
    const size_t *out_strides,     // Output tensor strides
    size_t out_size,               // Total number of output elements
    size_t nd                      // Number of dimensions
)
{
    // Maximum supported dimensions
    // const size_t MAX_DIMS = 10;

    // Each block handles one output element
    size_t out_idx = blockIdx.x;
    if (out_idx >= out_size)
        return;

    // Compute output coordinates from out_idx
    size_t out_coords[MAX_ND];
    size_t tmp = out_idx;
    for (ptrdiff_t d = nd - 1; d >= 0; --d)
    {
        out_coords[d] = tmp % out_shape[d];
        tmp /= out_shape[d];
    }

    // Compute output offset using out_strides
    size_t out_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        out_offset += out_coords[d] * out_strides[d];
    }

    // Compute base offset in the input tensor
    size_t base_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        size_t in_c = (out_shape[d] == 1) ? 0 : out_coords[d];
        base_offset += in_c * in_strides[d];
    }

    // Identify reduced dimensions
    size_t reduced_dims[MAX_ND];
    size_t reduced_sizes[MAX_ND];
    size_t num_reduced = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        if (out_shape[d] == 1 && in_shape[d] > 1)
        {
            reduced_dims[num_reduced] = d;
            reduced_sizes[num_reduced] = in_shape[d];
            num_reduced++;
        }
    }

    // Compute total number of elements to sum
    size_t N = 1;
    for (size_t j = 0; j < num_reduced; ++j)
    {
        N *= reduced_sizes[j];
    }

    // Each thread computes a partial sum
    __hip_bfloat16_raw partial_sum = (__hip_bfloat16_raw)0.0;
    for (size_t i = threadIdx.x; i < N; i += blockDim.x)
    {
        // Compute coordinates in the reduced dimensions from flattened index i
        size_t reduced_coords[MAX_ND];
        size_t tmp_i = i;
        for (ptrdiff_t j = num_reduced - 1; j >= 0; --j)
        {
            reduced_coords[j] = tmp_i % reduced_sizes[j];
            tmp_i /= reduced_sizes[j];
        }

        // Compute input offset
        size_t offset = base_offset;
        for (size_t j = 0; j < num_reduced; ++j)
        {
            size_t d = reduced_dims[j];
            offset += reduced_coords[j] * in_strides[d];
        }

        // Accumulate into partial sum
        partial_sum = partial_sum + d_in[offset];
    }

    // Use shared memory for reduction within the block
    extern __shared__ __hip_bfloat16_raw shared_sumsb[];
    shared_sumsb[threadIdx.x] = partial_sum;
    __syncthreads();

    // Perform parallel reduction in shared memory
    // Assumes blockDim.x is a power of two
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            shared_sumsb[threadIdx.x] = shared_sumsb[threadIdx.x] + shared_sumsb[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Thread 0 writes the final sum to the output
    if (threadIdx.x == 0)
    {
        d_out[out_offset] = shared_sumsb[0];
    }
}

__global__ void tomoSumToKernel(
    const float *d_in,         // Input tensor data
    const size_t *in_shape,    // Input tensor shape
    const size_t *in_strides,  // Input tensor strides
    float *d_out,              // Output tensor data
    const size_t *out_shape,   // Output tensor shape
    const size_t *out_strides, // Output tensor strides
    size_t out_size,           // Total number of output elements
    size_t nd                  // Number of dimensions
)
{
    // Maximum supported dimensions
    // const size_t MAX_DIMS = 10;

    // Each block handles one output element
    size_t out_idx = blockIdx.x;
    if (out_idx >= out_size)
        return;

    // Compute output coordinates from out_idx
    size_t out_coords[MAX_ND];
    size_t tmp = out_idx;
    for (ptrdiff_t d = nd - 1; d >= 0; --d)
    {
        out_coords[d] = tmp % out_shape[d];
        tmp /= out_shape[d];
    }

    // Compute output offset using out_strides
    size_t out_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        out_offset += out_coords[d] * out_strides[d];
    }

    // Compute base offset in the input tensor
    size_t base_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        size_t in_c = (out_shape[d] == 1) ? 0 : out_coords[d];
        base_offset += in_c * in_strides[d];
    }

    // Identify reduced dimensions
    size_t reduced_dims[MAX_ND];
    size_t reduced_sizes[MAX_ND];
    size_t num_reduced = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        if (out_shape[d] == 1 && in_shape[d] > 1)
        {
            reduced_dims[num_reduced] = d;
            reduced_sizes[num_reduced] = in_shape[d];
            num_reduced++;
        }
    }

    // Compute total number of elements to sum
    size_t N = 1;
    for (size_t j = 0; j < num_reduced; ++j)
    {
        N *= reduced_sizes[j];
    }

    // Each thread computes a partial sum
    float partial_sum = (float)0.0;
    for (size_t i = threadIdx.x; i < N; i += blockDim.x)
    {
        // Compute coordinates in the reduced dimensions from flattened index i
        size_t reduced_coords[MAX_ND];
        size_t tmp_i = i;
        for (ptrdiff_t j = num_reduced - 1; j >= 0; --j)
        {
            reduced_coords[j] = tmp_i % reduced_sizes[j];
            tmp_i /= reduced_sizes[j];
        }

        // Compute input offset
        size_t offset = base_offset;
        for (size_t j = 0; j < num_reduced; ++j)
        {
            size_t d = reduced_dims[j];
            offset += reduced_coords[j] * in_strides[d];
        }

        // Accumulate into partial sum
        partial_sum = partial_sum + d_in[offset];
    }

    // Use shared memory for reduction within the block
    extern __shared__ float shared_sumsf[];
    shared_sumsf[threadIdx.x] = partial_sum;
    __syncthreads();

    // Perform parallel reduction in shared memory
    // Assumes blockDim.x is a power of two
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            shared_sumsf[threadIdx.x] = shared_sumsf[threadIdx.x] + shared_sumsf[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Thread 0 writes the final sum to the output
    if (threadIdx.x == 0)
    {
        d_out[out_offset] = shared_sumsf[0];
    }
}

__global__ void tomoSumToKernel(
    const double *d_in,        // Input tensor data
    const size_t *in_shape,    // Input tensor shape
    const size_t *in_strides,  // Input tensor strides
    double *d_out,             // Output tensor data
    const size_t *out_shape,   // Output tensor shape
    const size_t *out_strides, // Output tensor strides
    size_t out_size,           // Total number of output elements
    size_t nd                  // Number of dimensions
)
{
    // Maximum supported dimensions
    // const size_t MAX_DIMS = 10;

    // Each block handles one output element
    size_t out_idx = blockIdx.x;
    if (out_idx >= out_size)
        return;

    // Compute output coordinates from out_idx
    size_t out_coords[MAX_ND];
    size_t tmp = out_idx;
    for (ptrdiff_t d = nd - 1; d >= 0; --d)
    {
        out_coords[d] = tmp % out_shape[d];
        tmp /= out_shape[d];
    }

    // Compute output offset using out_strides
    size_t out_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        out_offset += out_coords[d] * out_strides[d];
    }

    // Compute base offset in the input tensor
    size_t base_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        size_t in_c = (out_shape[d] == 1) ? 0 : out_coords[d];
        base_offset += in_c * in_strides[d];
    }

    // Identify reduced dimensions
    size_t reduced_dims[MAX_ND];
    size_t reduced_sizes[MAX_ND];
    size_t num_reduced = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        if (out_shape[d] == 1 && in_shape[d] > 1)
        {
            reduced_dims[num_reduced] = d;
            reduced_sizes[num_reduced] = in_shape[d];
            num_reduced++;
        }
    }

    // Compute total number of elements to sum
    size_t N = 1;
    for (size_t j = 0; j < num_reduced; ++j)
    {
        N *= reduced_sizes[j];
    }

    // Each thread computes a partial sum
    double partial_sum = (double)0.0;
    for (size_t i = threadIdx.x; i < N; i += blockDim.x)
    {
        // Compute coordinates in the reduced dimensions from flattened index i
        size_t reduced_coords[MAX_ND];
        size_t tmp_i = i;
        for (ptrdiff_t j = num_reduced - 1; j >= 0; --j)
        {
            reduced_coords[j] = tmp_i % reduced_sizes[j];
            tmp_i /= reduced_sizes[j];
        }

        // Compute input offset
        size_t offset = base_offset;
        for (size_t j = 0; j < num_reduced; ++j)
        {
            size_t d = reduced_dims[j];
            offset += reduced_coords[j] * in_strides[d];
        }

        // Accumulate into partial sum
        partial_sum = partial_sum + d_in[offset];
    }

    // Use shared memory for reduction within the block
    extern __shared__ double shared_sumsd[];
    shared_sumsd[threadIdx.x] = partial_sum;
    __syncthreads();

    // Perform parallel reduction in shared memory
    // Assumes blockDim.x is a power of two
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            shared_sumsd[threadIdx.x] = shared_sumsd[threadIdx.x] + shared_sumsd[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Thread 0 writes the final sum to the output
    if (threadIdx.x == 0)
    {
        d_out[out_offset] = shared_sumsd[0];
    }
}

template <typename T>
hipError_t tomoSumTo(
    const T *d_in,             // Input tensor data on device
    T *d_out,                  // Output tensor data on device
    size_t const *in_shape,    // Input tensor shape on host
    size_t in_shape_len,       // Length of in_shape
    size_t const *out_shape,   // Output tensor shape on host
    size_t out_shape_len,      // Length of out_shape
    size_t const *in_strides,  // Input tensor strides on host
    size_t in_strides_len,     // Length of in_strides
    size_t const *out_strides, // Output tensor strides on host
    size_t out_strides_len,    // Length of out_strides
    size_t in_size,            // Total size of input (unused)
    size_t out_size,           // Total size of output
    size_t nd,                 // Number of dimensions
    hipStream_t stream        // CUDA stream for asynchronous execution
)
{
    // Maximum supported dimensions
    const size_t MAX_DIMS = 10;

    // Validate inputs
    if (nd > MAX_DIMS || in_shape_len != nd || out_shape_len != nd ||
        in_strides_len != nd || out_strides_len != nd)
    {
        return hipErrorInvalidValue;
    }

    // Device buffers for shapes and strides
    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream));

    // Copy shapes and strides from host to device
    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));

    // Launch kernel
    const int threads = 256;          // Must be a power of two for reduction
    const int blocks = (int)out_size; // One block per output element
    const int shared_mem_size = (int)threads * (int)sizeof(T);
    tomoSumToKernel<<<blocks, threads, shared_mem_size, stream>>>(
        d_in, d_in_shape, d_in_strides, d_out, d_out_shape, d_out_strides, out_size, nd);

    // Check for kernel launch errors
    CHECK_CUDA(hipGetLastError());

    // Free device memory
    CHECK_CUDA(hipFreeAsync(d_in_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_out_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_in_strides, stream));
    CHECK_CUDA(hipFreeAsync(d_out_strides, stream));

    return hipSuccess;
}

#define BLOCK_SIZE 16

__global__ void tomoLinearKernelB(__hip_bfloat16_raw const *A, __hip_bfloat16_raw const *B, size_t M, size_t K, size_t N, __hip_bfloat16_raw const *bias, __hip_bfloat16_raw *C)
{
    auto row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    auto col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    auto sum = (__hip_bfloat16_raw)0.0;
    for (auto k = (size_t)0; k < K; k += BLOCK_SIZE)
    {
        __shared__ __hip_bfloat16_raw shared_a_b[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ __hip_bfloat16_raw shared_b_b[BLOCK_SIZE][BLOCK_SIZE];

        // Load Ashared
        if (row < M && k + threadIdx.x < K)
        {
            shared_a_b[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        }
        else
        {
            shared_a_b[threadIdx.y][threadIdx.x] = (__hip_bfloat16_raw)0.0;
        }

        // Load Bshared
        if (k + threadIdx.y < K && col < N)
        {
            shared_b_b[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        }
        else
        {
            shared_b_b[threadIdx.y][threadIdx.x] = (__hip_bfloat16_raw)0.0;
        }

        __syncthreads();

        for (auto m = (size_t)0; m < BLOCK_SIZE; m++)
        {
            if (k + m < K)
            {
                sum = sum + shared_a_b[threadIdx.y][m] * shared_b_b[m][threadIdx.x];
            }
        }

        __syncthreads();
    }
    if (row < M && col < N)
    {
        C[row * N + col] = sum;

        if (bias != NULL)
        {
            C[row * N + col] = C[row * N + col] + bias[row * N + col];
        }
    }
}

__global__ void tomoLinearKernelH(__half_raw const *A, __half_raw const *B, size_t M, size_t K, size_t N, __half_raw const *bias, __half_raw *C)
{
    auto row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    auto col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    auto sum = (__half_raw)0.0;
    for (auto k = (size_t)0; k < K; k += BLOCK_SIZE)
    {
        __shared__ __half_raw shared_a_h[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ __half_raw shared_b_h[BLOCK_SIZE][BLOCK_SIZE];

        // Load Ashared
        if (row < M && k + threadIdx.x < K)
        {
            shared_a_h[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        }
        else
        {
            shared_a_h[threadIdx.y][threadIdx.x] = (__half_raw)0.0;
        }

        // Load Bshared
        if (k + threadIdx.y < K && col < N)
        {
            shared_b_h[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        }
        else
        {
            shared_b_h[threadIdx.y][threadIdx.x] = (__half_raw)0.0;
        }

        __syncthreads();

        for (auto m = (size_t)0; m < BLOCK_SIZE; m++)
        {
            if (k + m < K)
            {
                sum = sum + shared_a_h[threadIdx.y][m] * shared_b_h[m][threadIdx.x];
            }
        }

        __syncthreads();
    }
    if (row < M && col < N)
    {
        C[row * N + col] = sum;

        if (bias != NULL)
        {
            C[row * N + col] = C[row * N + col] + bias[row * N + col];
        }
    }
}

__global__ void tomoLinearKernelF(float const *A, float const *B, size_t M, size_t K, size_t N, float const *bias, float *C)
{
    auto row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    auto col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    auto sum = 0.0f;
    for (auto k = (size_t)0; k < K; k += BLOCK_SIZE)
    {
        __shared__ float shared_a_f[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float shared_b_f[BLOCK_SIZE][BLOCK_SIZE];

        // Load Ashared
        if (row < M && k + threadIdx.x < K)
        {
            shared_a_f[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        }
        else
        {
            shared_a_f[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Load Bshared
        if (k + threadIdx.y < K && col < N)
        {
            shared_b_f[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        }
        else
        {
            shared_b_f[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (auto m = (size_t)0; m < BLOCK_SIZE; m++)
        {
            if (k + m < K)
            {
                sum += shared_a_f[threadIdx.y][m] * shared_b_f[m][threadIdx.x];
            }
        }

        __syncthreads();
    }
    if (row < M && col < N)
    {
        C[row * N + col] = sum;

        if (bias != NULL)
        {
            C[row * N + col] += bias[row * N + col];
        }
    }
}

__global__ void tomoLinearKernelD(double const *A, double const *B, size_t M, size_t K, size_t N, double const *bias, double *C)
{
    auto row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    auto col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    auto sum = 0.0;
    for (auto k = (size_t)0; k < K; k += BLOCK_SIZE)
    {
        __shared__ double shared_a_d[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double shared_b_d[BLOCK_SIZE][BLOCK_SIZE];

        // Load Ashared
        if (row < M && k + threadIdx.x < K)
        {
            shared_a_d[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        }
        else
        {
            shared_a_d[threadIdx.y][threadIdx.x] = 0.0;
        }

        // Load Bshared
        if (k + threadIdx.y < K && col < N)
        {
            shared_b_d[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        }
        else
        {
            shared_b_d[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        for (auto m = (size_t)0; m < BLOCK_SIZE; m++)
        {
            if (k + m < K)
            {
                sum += shared_a_d[threadIdx.y][m] * shared_b_d[m][threadIdx.x];
            }
        }

        __syncthreads();
    }
    if (row < M && col < N)
    {
        C[row * N + col] = sum;

        if (bias != NULL)
        {
            C[row * N + col] += bias[row * N + col];
        }
    }
}

template <typename T>
hipError_t tomoLinear(T const *A, T const *B, size_t M, size_t K, size_t N, T const *bias, T *C, hipStream_t stream)
{

    dim3 gridDim(((unsigned int)N + BLOCK_SIZE - 1) / BLOCK_SIZE, ((unsigned int)M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
    {
        tomoLinearKernelB<<<gridDim, blockDim, 0, stream>>>(A, B, M, K, N, bias, C);
    }
    else if constexpr (std::is_same_v<T, __half_raw>)
    {
        tomoLinearKernelH<<<gridDim, blockDim, 0, stream>>>(A, B, M, K, N, bias, C);
    }
    else if constexpr (std::is_same_v<T, float>)
    {
        tomoLinearKernelF<<<gridDim, blockDim, 0, stream>>>(A, B, M, K, N, bias, C);
    }
    else if constexpr (std::is_same_v<T, double>)
    {
        tomoLinearKernelD<<<gridDim, blockDim, 0, stream>>>(A, B, M, K, N, bias, C);
    }

    return hipGetLastError();
}

template <typename T>
struct TileDims
{
    // Default tile sizes for float, half, etc.
    static constexpr int BM = 64;
    static constexpr int BK = 32;
    static constexpr int BN = 64;
};

// Specialization for double: smaller BK to reduce shared memory usage
template <>
struct TileDims<double>
{
    static constexpr int BM = 64;
    static constexpr int BK = 16;
    static constexpr int BN = 64;
};
#define TM 4 // Sub-tile height per thread
#define TN 4 // Sub-tile width per thread

#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))

// static __device__ inline half2 vload_half2(const __half_raw* ptr)
// {
//     // If your environment supports it, you can do direct pointer casts:
//     // return *(reinterpret_cast<const half2*>(ptr));
//     // Or an inline PTX approach. For brevity, let's just do:
//     return __halves2half2(ptr[0], ptr[1]);
// }

__global__ void tomoLinearKernelimpH(
    const __half_raw *__restrict__ A, // (M x K)
    const __half_raw *__restrict__ B, // (K x N)
    size_t M, size_t K, size_t N,
    const __half_raw *__restrict__ bias, // (optional, M x N)
    __half_raw *__restrict__ C           // (M x N)
)
{
    constexpr int BM = TileDims<__half_raw>::BM;
    constexpr int BK = TileDims<__half_raw>::BK;
    constexpr int BN = TileDims<__half_raw>::BN;

    // 2D index in the grid
    const int blockRow = blockIdx.y; // which block in dimension M
    const int blockCol = blockIdx.x; // which block in dimension N

    // 2D index in the block
    const int threadRow = threadIdx.y;
    const int threadCol = threadIdx.x;

    // Global row/column for the block
    const int globalRow = blockRow * BM;
    const int globalCol = blockCol * BN;

    // Start offsets in shared memory
    // We'll keep two buffers for double‐buffering: As[2][BM][BK], Bs[2][BK][BN].
    __shared__ __half_raw As[2][BM][BK];
    __shared__ __half_raw Bs[2][BK][BN];

    // Each thread accumulates a sub‐tile of size TM x TN in registers
    // i.e. 4×4, or 4×8, etc.
    __half_raw sum[TM][TN];
#pragma unroll
    for (int i = 0; i < TM; i++)
    {
#pragma unroll
        for (int j = 0; j < TN; j++)
        {
            sum[i][j] = __half_raw(0.0f);
        }
    }

    // We’ll assume (BM * BK) and (BK * BN) are divisible by (blockDim.x * blockDim.y).
    // So each thread can copy multiple elements in a loop.

    // Compute how many tiles along K
    const int numTiles = (K + BK - 1) / BK; // assume K multiple of BK for simplicity

    // For double buffering, we keep track of the “read index” vs. the “compute index”
    int loadIdx = 0; // we load into this buffer
    int compIdx = 1; // we compute from this buffer

    // Local thread id if we want 1D
    int tid = threadRow * blockDim.x + threadCol;
    int tdim = blockDim.x * blockDim.y;

    //----------- FIRST TILE LOAD (Tile 0) -----------
    {
        // load A tile 0 into As[loadIdx], B tile 0 into Bs[loadIdx]
        // Each thread copies multiple elements from global to shared

        // A tile is (BM×BK)
        for (int idx = tid; idx < BM * BK; idx += tdim)
        {
            int a_row = idx / BK;       // local row
            int a_col = idx % BK;       // local col
            int ga = globalRow + a_row; // global row in A
            int ka = 0 + a_col;         // global col in A (tile 0 => offset=0)
            // Check boundary
            if (ga < M && ka < K)
            {
                As[loadIdx][a_row][a_col] = A[ga * K + ka];
            }
            else
            {
                As[loadIdx][a_row][a_col] = __half_raw(0.0f);
            }
        }
        // B tile is (BK×BN)
        for (int idx = tid; idx < BK * BN; idx += tdim)
        {
            int b_row = idx / BN;
            int b_col = idx % BN;
            int kb = 0 + b_row; // global row in B (tile 0 => offset=0)
            int gb = globalCol + b_col;
            if (kb < K && gb < N)
            {
                Bs[loadIdx][b_row][b_col] = B[kb * N + gb];
            }
            else
            {
                Bs[loadIdx][b_row][b_col] = __half_raw(0.0f);
            }
        }
    }
    __syncthreads();

    //----------- MAIN LOOP OVER TILES -----------
    for (int tile = 0; tile < numTiles; tile++)
    {
        // Swap loadIdx and compIdx so we compute on the tile that was loaded
        loadIdx = tile % 2;
        compIdx = (tile + 1) % 2;

        // Launch asynchronous load for the *next* tile if not the last tile
        if (tile < numTiles - 1)
        {
            int nextOffset = (tile + 1) * BK; // starting K for next tile

            // A tile in the next offset
            for (int idx = tid; idx < BM * BK; idx += tdim)
            {
                int a_row = idx / BK;
                int a_col = idx % BK;
                int ga = globalRow + a_row;
                int ka = nextOffset + a_col;
                if (ga < M && ka < K)
                {
                    As[compIdx][a_row][a_col] = A[ga * K + ka];
                }
                else
                {
                    As[compIdx][a_row][a_col] = __half_raw(0.0f);
                }
            }
            // B tile in the next offset
            for (int idx = tid; idx < BK * BN; idx += tdim)
            {
                int b_row = idx / BN;
                int b_col = idx % BN;
                int kb = nextOffset + b_row;
                int gb = globalCol + b_col;
                if (kb < K && gb < N)
                {
                    Bs[compIdx][b_row][b_col] = B[kb * N + gb];
                }
                else
                {
                    Bs[compIdx][b_row][b_col] = __half_raw(0.0f);
                }
            }
        }

        // We must __syncthreads() before we start computing on As[loadIdx], Bs[loadIdx]
        __syncthreads();

        //----- Multiply-Accumulate on the current tile "tile" from As[loadIdx], Bs[loadIdx] -----
        // Each tile is BK wide in K dimension
        // We can do partial unrolling (hand unroll the loop over BK).
        // For simplicity, we do #pragma unroll in a normal for loop:

#pragma unroll
        for (int m = 0; m < BK; m++)
        {
            // Each thread loads a tiny sub-vector from shared
            __half_raw aFrag[TM];
            __half_raw bFrag[TN];

#pragma unroll
            for (int i = 0; i < TM; i++)
            {
                int r = threadRow * TM + i; // local row in the BM dimension
                aFrag[i] = As[loadIdx][r][m];
            }
#pragma unroll
            for (int j = 0; j < TN; j++)
            {
                int c = threadCol * TN + j; // local col in the BN dimension
                bFrag[j] = Bs[loadIdx][m][c];
            }

// Now do the dot products: sum[i][j] += aFrag[i] * bFrag[j]
#pragma unroll
            for (int i = 0; i < TM; i++)
            {
#pragma unroll
                for (int j = 0; j < TN; j++)
                {
                    sum[i][j] = __hfma(aFrag[i], bFrag[j], sum[i][j]);
                }
            }
        }
        __syncthreads(); // ensure we've finished compute before overwriting As/Bs in next iteration
    } // end for tile

//----------- Write the results to global memory -----------
// plus the bias if non-null
#pragma unroll
    for (int i = 0; i < TM; i++)
    {
        int rowOut = globalRow + threadRow * TM + i;
        if (rowOut < M)
        {
#pragma unroll
            for (int j = 0; j < TN; j++)
            {
                int colOut = globalCol + threadCol * TN + j;
                if (colOut < N)
                {
                    size_t outIdx = rowOut * N + colOut;
                    __half_raw val = sum[i][j];
                    if (bias != nullptr)
                    {
                        val = __hadd(val, bias[outIdx]);
                    }
                    C[outIdx] = val;
                }
            }
        }
    }
}

__device__ inline __hip_bfloat16 bf16_fma(__hip_bfloat16 a, __hip_bfloat16 b, __hip_bfloat16 c)
{
    // Convert to float
    float fa = __bfloat162float(a);
    float fb = __bfloat162float(b);
    float fc = __bfloat162float(c);
    // Perform FMA in float
    float fres = fa * fb + fc;
    // Convert back
    return __float2bfloat16_rn(fres);
}

// The double‐buffered bfloat16 kernel
__global__ void tomoLinearKernelimpB(
    const __hip_bfloat16_raw *__restrict__ A, // (M x K)
    const __hip_bfloat16_raw *__restrict__ B, // (K x N)
    size_t M, size_t K, size_t N,
    const __hip_bfloat16_raw *__restrict__ bias, // (optional, M x N)
    __hip_bfloat16_raw *__restrict__ C           // (M x N)
)
{

    constexpr int BM = TileDims<__hip_bfloat16_raw>::BM;
    constexpr int BK = TileDims<__hip_bfloat16_raw>::BK;
    constexpr int BN = TileDims<__hip_bfloat16_raw>::BN;

    // 2D block index
    const int blockRow = blockIdx.y;
    const int blockCol = blockIdx.x;

    // 2D thread index
    const int threadRow = threadIdx.y;
    const int threadCol = threadIdx.x;

    // Start row/col in global memory
    const int globalRow = blockRow * BM;
    const int globalCol = blockCol * BN;

    // Double‐buffered shared memory
    __shared__ __hip_bfloat16_raw As[2][BM][BK];
    __shared__ __hip_bfloat16_raw Bs[2][BK][BN];

    // Accumulator sub‐tile in registers (TM × TN)
    __hip_bfloat16_raw sum[TM][TN];
#pragma unroll
    for (int i = 0; i < TM; i++)
    {
#pragma unroll
        for (int j = 0; j < TN; j++)
        {
            sum[i][j] = (__hip_bfloat16_raw)0.0f;
        }
    }

    const int numTiles = (K + BK - 1) / BK;
    // 1D thread id for loading
    int tid = threadRow * blockDim.x + threadCol;
    int tdim = blockDim.x * blockDim.y;

    int loadIdx = 0;
    int compIdx = 1;

    //---------- Load first tile (tile=0) ----------
    {
        // A tile (BM×BK)
        for (int idx = tid; idx < BM * BK; idx += tdim)
        {
            int a_row = idx / BK;
            int a_col = idx % BK;
            int ga = globalRow + a_row;
            int ka = a_col;
            if (ga < M && ka < K)
            {
                As[loadIdx][a_row][a_col] = A[ga * K + ka];
            }
            else
            {
                As[loadIdx][a_row][a_col] = (__hip_bfloat16_raw)0.0f;
            }
        }
        // B tile (BK×BN)
        for (int idx = tid; idx < BK * BN; idx += tdim)
        {
            int b_row = idx / BN;
            int b_col = idx % BN;
            int kb = b_row;
            int gb = globalCol + b_col;
            if (kb < K && gb < N)
            {
                Bs[loadIdx][b_row][b_col] = B[kb * N + gb];
            }
            else
            {
                Bs[loadIdx][b_row][b_col] = (__hip_bfloat16_raw)0.0f;
            }
        }
    }
    __syncthreads();

    //---------- Main loop over K tiles ----------
    for (int tile = 0; tile < numTiles; tile++)
    {
        loadIdx = tile % 2;
        compIdx = (tile + 1) % 2;

        // Launch load for the next tile
        if (tile < numTiles - 1)
        {
            int nextOffset = (tile + 1) * BK;
            // Load As[compIdx], Bs[compIdx]
            for (int idx = tid; idx < BM * BK; idx += tdim)
            {
                int a_row = idx / BK;
                int a_col = idx % BK;
                int ga = globalRow + a_row;
                int ka = nextOffset + a_col;
                if (ga < M && ka < K)
                {
                    As[compIdx][a_row][a_col] = A[ga * K + ka];
                }
                else
                {
                    As[compIdx][a_row][a_col] = (__hip_bfloat16_raw)0.0f;
                }
            }
            for (int idx = tid; idx < BK * BN; idx += tdim)
            {
                int b_row = idx / BN;
                int b_col = idx % BN;
                int kb = nextOffset + b_row;
                int gb = globalCol + b_col;
                if (kb < K && gb < N)
                {
                    Bs[compIdx][b_row][b_col] = B[kb * N + gb];
                }
                else
                {
                    Bs[compIdx][b_row][b_col] = (__hip_bfloat16_raw)0.0f;
                }
            }
        }
        __syncthreads();

// Multiply-accumulate on tile "tile" in As[loadIdx], Bs[loadIdx]
#pragma unroll
        for (int m = 0; m < BK; m++)
        {
            __hip_bfloat16_raw aFrag[TM];
            __hip_bfloat16_raw bFrag[TN];
#pragma unroll
            for (int i = 0; i < TM; i++)
            {
                int r = threadRow * TM + i;
                aFrag[i] = As[loadIdx][r][m];
            }
#pragma unroll
            for (int j = 0; j < TN; j++)
            {
                int c = threadCol * TN + j;
                bFrag[j] = Bs[loadIdx][m][c];
            }
// Accumulate
#pragma unroll
            for (int i = 0; i < TM; i++)
            {
#pragma unroll
                for (int j = 0; j < TN; j++)
                {
                    sum[i][j] = bf16_fma(aFrag[i], bFrag[j], sum[i][j]);
                }
            }
        }
        __syncthreads();
    }

//----------- Write to global memory (+ optional bias) -----------
#pragma unroll
    for (int i = 0; i < TM; i++)
    {
        int rowOut = globalRow + threadRow * TM + i;
        if (rowOut < M)
        {
#pragma unroll
            for (int j = 0; j < TN; j++)
            {
                int colOut = globalCol + threadCol * TN + j;
                if (colOut < N)
                {
                    size_t outIdx = rowOut * N + colOut;
                    __hip_bfloat16_raw val = sum[i][j];
                    if (bias != nullptr)
                    {
                        val = val + bias[outIdx];
                    }
                    C[outIdx] = val;
                }
            }
        }
    }
}

__device__ inline float floatFMA(float a, float b, float c)
{
    // If your GPU supports single-precision fused multiply add by default,
    // you can do:
    return __fmaf_rn(a, b, c);
    // or simply:
    // return a * b + c;
}

// Double‐buffered float kernel
__global__ void tomoLinearKernelimpF(
    const float *__restrict__ A,
    const float *__restrict__ B,
    size_t M, size_t K, size_t N,
    const float *__restrict__ bias,
    float *__restrict__ C)
{

    constexpr int BM = TileDims<float>::BM;
    constexpr int BK = TileDims<float>::BK;
    constexpr int BN = TileDims<float>::BN;

    const int blockRow = blockIdx.y;
    const int blockCol = blockIdx.x;
    const int threadRow = threadIdx.y;
    const int threadCol = threadIdx.x;

    const int globalRow = blockRow * BM;
    const int globalCol = blockCol * BN;

    __shared__ float As[2][BM][BK];
    __shared__ float Bs[2][BK][BN];

    float sum[TM][TN];
#pragma unroll
    for (int i = 0; i < TM; i++)
    {
#pragma unroll
        for (int j = 0; j < TN; j++)
        {
            sum[i][j] = 0.0f;
        }
    }

    const int numTiles = (K + BK - 1) / BK;
    int tid = threadRow * blockDim.x + threadCol;
    int tdim = blockDim.x * blockDim.y;

    int loadIdx = 0;
    int compIdx = 1;

    // Load tile 0
    {
        for (int idx = tid; idx < BM * BK; idx += tdim)
        {
            int a_row = idx / BK;
            int a_col = idx % BK;
            int ga = globalRow + a_row;
            int ka = a_col;
            As[loadIdx][a_row][a_col] = (ga < M && ka < K) ? A[ga * K + ka] : 0.0f;
        }
        for (int idx = tid; idx < BK * BN; idx += tdim)
        {
            int b_row = idx / BN;
            int b_col = idx % BN;
            int kb = b_row;
            int gb = globalCol + b_col;
            Bs[loadIdx][b_row][b_col] = (kb < K && gb < N) ? B[kb * N + gb] : 0.0f;
        }
    }
    __syncthreads();

    // main loop
    for (int tile = 0; tile < numTiles; tile++)
    {
        loadIdx = tile % 2;
        compIdx = (tile + 1) % 2;

        if (tile < numTiles - 1)
        {
            int nextOffset = (tile + 1) * BK;
            for (int idx = tid; idx < BM * BK; idx += tdim)
            {
                int a_row = idx / BK;
                int a_col = idx % BK;
                int ga = globalRow + a_row;
                int ka = nextOffset + a_col;
                As[compIdx][a_row][a_col] =
                    (ga < M && ka < K) ? A[ga * K + ka] : 0.0f;
            }
            for (int idx = tid; idx < BK * BN; idx += tdim)
            {
                int b_row = idx / BN;
                int b_col = idx % BN;
                int kb = nextOffset + b_row;
                int gb = globalCol + b_col;
                Bs[compIdx][b_row][b_col] =
                    (kb < K && gb < N) ? B[kb * N + gb] : 0.0f;
            }
        }
        __syncthreads();

// Multiply-accumulate
#pragma unroll
        for (int m = 0; m < BK; m++)
        {
            float aFrag[TM];
            float bFrag[TN];
#pragma unroll
            for (int i = 0; i < TM; i++)
            {
                int r = threadRow * TM + i;
                aFrag[i] = As[loadIdx][r][m];
            }
#pragma unroll
            for (int j = 0; j < TN; j++)
            {
                int c = threadCol * TN + j;
                bFrag[j] = Bs[loadIdx][m][c];
            }
#pragma unroll
            for (int i = 0; i < TM; i++)
            {
#pragma unroll
                for (int j = 0; j < TN; j++)
                {
                    sum[i][j] = floatFMA(aFrag[i], bFrag[j], sum[i][j]);
                }
            }
        }
        __syncthreads();
    }

// Write output
#pragma unroll
    for (int i = 0; i < TM; i++)
    {
        int rowOut = globalRow + threadRow * TM + i;
        if (rowOut < M)
        {
#pragma unroll
            for (int j = 0; j < TN; j++)
            {
                int colOut = globalCol + threadCol * TN + j;
                if (colOut < N)
                {
                    size_t outIdx = rowOut * N + colOut;
                    float val = sum[i][j];
                    if (bias != nullptr)
                    {
                        val = val + bias[outIdx];
                    }
                    C[outIdx] = val;
                }
            }
        }
    }
}

__device__ inline double doubleFMA(double a, double b, double c)
{
    // If your hardware or compiler supports double-precision FMA by default,
    // you can do:
    return a * b + c;
    // or fma(a, b, c).
}

// Double-buffered double kernel
__global__ void tomoLinearKernelimpD(
    const double *__restrict__ A,
    const double *__restrict__ B,
    size_t M, size_t K, size_t N,
    const double *__restrict__ bias,
    double *__restrict__ C)
{
    constexpr int BM = TileDims<double>::BM;
    constexpr int BK = TileDims<double>::BK;
    constexpr int BN = TileDims<double>::BN;

    const int blockRow = blockIdx.y;
    const int blockCol = blockIdx.x;
    const int threadRow = threadIdx.y;
    const int threadCol = threadIdx.x;

    const int globalRow = blockRow * BM;
    const int globalCol = blockCol * BN;

    __shared__ double As[2][BM][BK];
    __shared__ double Bs[2][BK][BN];

    double sum[TM][TN];
#pragma unroll
    for (int i = 0; i < TM; i++)
    {
#pragma unroll
        for (int j = 0; j < TN; j++)
        {
            sum[i][j] = 0.0;
        }
    }

    const int numTiles = (K + BK - 1) / BK;
    int tid = threadRow * blockDim.x + threadCol;
    int tdim = blockDim.x * blockDim.y;

    int loadIdx = 0;
    int compIdx = 1;

    // First tile load
    {
        for (int idx = tid; idx < BM * BK; idx += tdim)
        {
            int a_row = idx / BK;
            int a_col = idx % BK;
            int ga = globalRow + a_row;
            int ka = a_col;
            As[loadIdx][a_row][a_col] =
                (ga < M && ka < K) ? A[ga * K + ka] : 0.0;
        }
        for (int idx = tid; idx < BK * BN; idx += tdim)
        {
            int b_row = idx / BN;
            int b_col = idx % BN;
            int kb = b_row;
            int gb = globalCol + b_col;
            Bs[loadIdx][b_row][b_col] =
                (kb < K && gb < N) ? B[kb * N + gb] : 0.0;
        }
    }
    __syncthreads();

    // Main loop
    for (int tile = 0; tile < numTiles; tile++)
    {
        loadIdx = tile % 2;
        compIdx = (tile + 1) % 2;

        // Next tile prefetch
        if (tile < numTiles - 1)
        {
            int nextOffset = (tile + 1) * BK;
            for (int idx = tid; idx < BM * BK; idx += tdim)
            {
                int a_row = idx / BK;
                int a_col = idx % BK;
                int ga = globalRow + a_row;
                int ka = nextOffset + a_col;
                As[compIdx][a_row][a_col] =
                    (ga < M && ka < K) ? A[ga * K + ka] : 0.0;
            }
            for (int idx = tid; idx < BK * BN; idx += tdim)
            {
                int b_row = idx / BN;
                int b_col = idx % BN;
                int kb = nextOffset + b_row;
                int gb = globalCol + b_col;
                Bs[compIdx][b_row][b_col] =
                    (kb < K && gb < N) ? B[kb * N + gb] : 0.0;
            }
        }
        __syncthreads();

// Multiply-accumulate
#pragma unroll
        for (int m = 0; m < BK; m++)
        {
            double aFrag[TM];
            double bFrag[TN];
#pragma unroll
            for (int i = 0; i < TM; i++)
            {
                int r = threadRow * TM + i;
                aFrag[i] = As[loadIdx][r][m];
            }
#pragma unroll
            for (int j = 0; j < TN; j++)
            {
                int c = threadCol * TN + j;
                bFrag[j] = Bs[loadIdx][m][c];
            }
#pragma unroll
            for (int i = 0; i < TM; i++)
            {
#pragma unroll
                for (int j = 0; j < TN; j++)
                {
                    sum[i][j] = doubleFMA(aFrag[i], bFrag[j], sum[i][j]);
                }
            }
        }
        __syncthreads();
    }

// Write out
#pragma unroll
    for (int i = 0; i < TM; i++)
    {
        int rowOut = globalRow + threadRow * TM + i;
        if (rowOut < M)
        {
#pragma unroll
            for (int j = 0; j < TN; j++)
            {
                int colOut = globalCol + threadCol * TN + j;
                if (colOut < N)
                {
                    size_t outIdx = rowOut * N + colOut;
                    double val = sum[i][j];
                    if (bias != nullptr)
                    {
                        val += bias[outIdx];
                    }
                    C[outIdx] = val;
                }
            }
        }
    }
}

template <typename T>
hipError_t tomoLinearImp(T const *A, T const *B, size_t M, size_t K, size_t N, T const *bias, T *C, hipStream_t stream)
{
    constexpr int BM = TileDims<T>::BM;
    // constexpr int BK = TileDims<T>::BK;
    constexpr int BN = TileDims<T>::BN;

    dim3 blockDim(BN / TN, BM / TM); // e.g., (16, 16) with BN=128, BM=128, TN=8, TM=8
    dim3 gridDim((unsigned int)CEIL_DIV(N, BN), (unsigned int)CEIL_DIV(M, BM));

    if constexpr (std::is_same_v<T, __half_raw>)
    {
        tomoLinearKernelimpH<<<gridDim, blockDim, 0, stream>>>(
            A, B, M, K, N, bias, C);
    }
    else if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
    {
        tomoLinearKernelimpB<<<gridDim, blockDim, 0, stream>>>(
            A, B, M, K, N, bias, C);
    }
    else if constexpr (std::is_same_v<T, float>)
    {
        tomoLinearKernelimpF<<<gridDim, blockDim, 0, stream>>>(
            A, B, M, K, N, bias, C);
    }
    else if constexpr (std::is_same_v<T, double>)
    {
        tomoLinearKernelimpD<<<gridDim, blockDim, 0, stream>>>(
            A, B, M, K, N, bias, C);
    }

    return hipGetLastError();
}

__global__ void tomoTransposeKernelH(const __half_raw *A, size_t M, size_t N, __half_raw *C)
{
    __shared__ __half_raw tile_h[BLOCK_SIZE][BLOCK_SIZE + 1];

    // (row, col) in the original A
    size_t row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    size_t col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // 1) Read input => shared memory
    if (row < M && col < N)
        tile_h[threadIdx.y][threadIdx.x] = A[row * N + col];
    else
        tile_h[threadIdx.y][threadIdx.x] = (__half_raw)0.0;

    __syncthreads();

    // (row, col) in transposed output (swap blockIdx.x, blockIdx.y)
    size_t new_row = blockIdx.x * BLOCK_SIZE + threadIdx.y;
    size_t new_col = blockIdx.y * BLOCK_SIZE + threadIdx.x;

    // 2) Write from shared memory => output
    if (new_row < N && new_col < M)
        C[new_row * M + new_col] = tile_h[threadIdx.x][threadIdx.y];
}

__global__ void tomoTransposeKernelB(const __hip_bfloat16_raw *A, size_t M, size_t N, __hip_bfloat16_raw *C)
{
    __shared__ __hip_bfloat16_raw tile_b[BLOCK_SIZE][BLOCK_SIZE + 1];

    size_t row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    size_t col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (row < M && col < N)
        tile_b[threadIdx.y][threadIdx.x] = A[row * N + col];
    else
        tile_b[threadIdx.y][threadIdx.x] = (__hip_bfloat16_raw)0.0;

    __syncthreads();

    size_t new_row = blockIdx.x * BLOCK_SIZE + threadIdx.y;
    size_t new_col = blockIdx.y * BLOCK_SIZE + threadIdx.x;

    if (new_row < N && new_col < M)
        C[new_row * M + new_col] = tile_b[threadIdx.x][threadIdx.y];
}

__global__ void tomoTransposeKernelF(const float *A, size_t M, size_t N, float *C)
{
    __shared__ float tile_f[BLOCK_SIZE][BLOCK_SIZE + 1];

    size_t row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    size_t col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (row < M && col < N)
        tile_f[threadIdx.y][threadIdx.x] = A[row * N + col];
    else
        tile_f[threadIdx.y][threadIdx.x] = 0.0f;

    __syncthreads();

    size_t new_row = blockIdx.x * BLOCK_SIZE + threadIdx.y;
    size_t new_col = blockIdx.y * BLOCK_SIZE + threadIdx.x;

    if (new_row < N && new_col < M)
        C[new_row * M + new_col] = tile_f[threadIdx.x][threadIdx.y];
}

__global__ void tomoTransposeKernelD(const double *A, size_t M, size_t N, double *C)
{
    __shared__ double tile_d[BLOCK_SIZE][BLOCK_SIZE + 1];

    size_t row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    size_t col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (row < M && col < N)
        tile_d[threadIdx.y][threadIdx.x] = A[row * N + col];
    else
        tile_d[threadIdx.y][threadIdx.x] = 0.0; // double literal

    __syncthreads();

    size_t new_row = blockIdx.x * BLOCK_SIZE + threadIdx.y;
    size_t new_col = blockIdx.y * BLOCK_SIZE + threadIdx.x;

    if (new_row < N && new_col < M)
        C[new_row * M + new_col] = tile_d[threadIdx.x][threadIdx.y];
}

// __global__ void tomoTransposeKernelH(__half_raw const *A, size_t M, size_t N, __half_raw *C)
// {
//     // Shared memory to hold a tile of the input matrix
//     __shared__ __half_raw tile_h[BLOCK_SIZE][BLOCK_SIZE + 1]; // +1 to avoid bank conflicts

//     // Input coordinates (reading from A)
//     auto x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
//     auto y = blockIdx.y * BLOCK_SIZE + threadIdx.y;

//     // Load data into shared memory (coalesced read from A)
//     if (y < M && x < N)
//     {
//         tile_h[threadIdx.y][threadIdx.x] = A[y * N + x];
//     }
//     else
//     {
//         tile_h[threadIdx.y][threadIdx.x] = (__half_raw)0.0; // Padding for out-of-bounds
//     }

//     __syncthreads();

//     // Output coordinates (writing to C)
//     auto tx = blockIdx.y * BLOCK_SIZE + threadIdx.x; // Swapped block indices
//     auto ty = blockIdx.x * BLOCK_SIZE + threadIdx.y;

//     // Write transposed data to global memory (coalesced write to C)
//     if (ty < N && tx < M)
//     {
//         C[tx * N + ty] = tile_h[threadIdx.x][threadIdx.y]; // Note swapped indices
//     }
// }

template <typename T>
hipError_t tomoTranspose(T const *A, size_t M, size_t N, T *C, hipStream_t stream)
{

    // dim3 gridDim(((unsigned int)N + BLOCK_SIZE - 1) / BLOCK_SIZE, ((unsigned int)M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    // dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(((unsigned int)N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 ((unsigned int)M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
    {
        tomoTransposeKernelB<<<gridDim, blockDim, 0, stream>>>(A, M, N, C);
    }
    else if constexpr (std::is_same_v<T, __half_raw>)
    {
        tomoTransposeKernelH<<<gridDim, blockDim, 0, stream>>>(A, M, N, C);
    }
    else if constexpr (std::is_same_v<T, float>)
    {
        tomoTransposeKernelF<<<gridDim, blockDim, 0, stream>>>(A, M, N, C);
    }
    else if constexpr (std::is_same_v<T, double>)
    {
        tomoTransposeKernelD<<<gridDim, blockDim, 0, stream>>>(A, M, N, C);
    }

    return hipGetLastError();
}

template <typename T>
__global__ void tomoMaxToKernel(
    T const *d_in, T *d_out,
    size_t const *d_in_shape, size_t const *d_out_shape,
    size_t const *d_in_strides, size_t const *d_out_strides,
    size_t in_size, size_t out_size, size_t nd)
{
    size_t out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
        return;

    // Unravel out_idx to out_coords
    size_t out_coords[MAX_ND];
    size_t tmp = out_idx;
    for (ptrdiff_t d = nd - 1; d >= 0; --d)
    {
        size_t dim_size = d_out_shape[d];
        out_coords[d] = tmp % dim_size;
        tmp /= dim_size;
    }

    // Initialize max_val to the smallest possible value for type T
    T max_val = std::numeric_limits<T>::lowest();
    size_t in_coords[MAX_ND];
    for (size_t in_idx = 0; in_idx < in_size; ++in_idx)
    {
        size_t unravel = in_idx;
        bool matches = true;
        for (ptrdiff_t d = nd - 1; d >= 0; --d)
        {
            size_t dim_size = d_in_shape[d];
            in_coords[d] = unravel % dim_size;
            unravel /= dim_size;
            if (d_out_shape[d] != 1)
            {
                size_t out_c = out_coords[d];
                size_t in_c = (d_in_shape[d] == 1) ? 0 : in_coords[d];
                if (out_c != in_c)
                {
                    matches = false;
                    break;
                }
            }
        }
        if (matches)
        {
            size_t in_offset = 0;
            for (size_t d = 0; d < nd; ++d)
            {
                in_offset += in_coords[d] * d_in_strides[d];
            }
            max_val = std::max(max_val, d_in[in_offset]);
        }
    }
    d_out[out_idx] = max_val;
}

template <typename T>
hipError_t tomoMaxTo(
    const T *d_in, T *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size, size_t out_size, size_t nd,
    hipStream_t stream)
{
    // Validate inputs
    if (out_strides_len != nd)
        return hipErrorInvalidValue;

    // Device buffers
    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream));

    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));

    // Launch kernel
    const int threads = 256;
    const int blocks = ((int)out_size + threads - 1) / threads;
    tomoMaxToKernel<<<blocks, threads, 0, stream>>>(
        d_in, d_out, d_in_shape, d_out_shape, d_in_strides, d_out_strides,
        in_size, out_size, nd);

    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipFreeAsync(d_in_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_out_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_in_strides, stream));
    CHECK_CUDA(hipFreeAsync(d_out_strides, stream));
    return hipSuccess;
}

template <typename T>
__global__ void tomoMinToKernel(
    T const *d_in, T *d_out,
    size_t const *d_in_shape, size_t const *d_out_shape,
    size_t const *d_in_strides, size_t const *d_out_strides,
    size_t in_size, size_t out_size, size_t nd)
{
    size_t out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
        return;

    // Unravel out_idx to out_coords
    size_t out_coords[MAX_ND];
    size_t tmp = out_idx;
    for (ptrdiff_t d = nd - 1; d >= 0; --d)
    {
        size_t dim_size = d_out_shape[d];
        out_coords[d] = tmp % dim_size;
        tmp /= dim_size;
    }

    // Initialize min_val to the largest possible value for type T
    T min_val = std::numeric_limits<T>::max();
    size_t in_coords[MAX_ND];
    for (size_t in_idx = 0; in_idx < in_size; ++in_idx)
    {
        size_t unravel = in_idx;
        bool matches = true;
        for (ptrdiff_t d = nd - 1; d >= 0; --d)
        {
            size_t dim_size = d_in_shape[d];
            in_coords[d] = unravel % dim_size;
            unravel /= dim_size;
            if (d_out_shape[d] != 1)
            {
                size_t out_c = out_coords[d];
                size_t in_c = (d_in_shape[d] == 1) ? 0 : in_coords[d];
                if (out_c != in_c)
                {
                    matches = false;
                    break;
                }
            }
        }
        if (matches)
        {
            size_t in_offset = 0;
            for (size_t d = 0; d < nd; ++d)
            {
                in_offset += in_coords[d] * d_in_strides[d];
            }
            min_val = std::min(min_val, d_in[in_offset]);
        }
    }
    d_out[out_idx] = min_val;
}

template <typename T>
hipError_t tomoMinTo(
    const T *d_in, T *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size, size_t out_size, size_t nd,
    hipStream_t stream)
{
    // Validate inputs
    if (out_strides_len != nd)
        return hipErrorInvalidValue;

    // Device buffers
    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream));

    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));

    // Launch kernel
    const int threads = 256;
    const int blocks = ((int)out_size + threads - 1) / threads;
    tomoMinToKernel<<<blocks, threads, 0, stream>>>(
        d_in, d_out, d_in_shape, d_out_shape, d_in_strides, d_out_strides,
        in_size, out_size, nd);

    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipFreeAsync(d_in_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_out_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_in_strides, stream));
    CHECK_CUDA(hipFreeAsync(d_out_strides, stream));
    return hipSuccess;
}

template <typename T>
__global__ void tomoTensordotKernel(
    const T *d_a, const T *d_b, T *d_out,
    const size_t *d_a_shape, const size_t *d_b_shape, const size_t *d_out_shape,
    const size_t *d_a_strides, const size_t *d_b_strides, const size_t *d_out_strides,
    size_t a_nd, size_t b_nd, size_t out_nd,
    size_t out_size, size_t num_contracted,
    const size_t *d_contracted_axes_a, const size_t *d_contracted_axes_b)
{
    size_t out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
        return;

    // Unravel output index
    size_t out_coords[MAX_ND];
    size_t tmp = out_idx;
    for (size_t d = out_nd; d-- > 0;)
    {
        out_coords[d] = tmp % d_out_shape[d];
        tmp /= d_out_shape[d];
    }

    // Initialize coordinates for A and B
    size_t a_coords[MAX_ND] = {0};
    size_t b_coords[MAX_ND] = {0};

    // Map output coordinates to non-contracted axes
    size_t a_non_contracted_idx = 0;
    size_t b_non_contracted_idx = 0;

    for (size_t d = 0; d < a_nd; ++d)
    {
        bool is_contracted = false;
        for (size_t p = 0; p < num_contracted; ++p)
        {
            if (d == d_contracted_axes_a[p])
            {
                is_contracted = true;
                break;
            }
        }
        if (!is_contracted)
        {
            a_coords[d] = out_coords[a_non_contracted_idx++];
        }
    }

    for (size_t d = 0; d < b_nd; ++d)
    {
        bool is_contracted = false;
        for (size_t p = 0; p < num_contracted; ++p)
        {
            if (d == d_contracted_axes_b[p])
            {
                is_contracted = true;
                break;
            }
        }
        if (!is_contracted)
        {
            b_coords[d] = out_coords[a_non_contracted_idx + b_non_contracted_idx++];
        }
    }

    // Contracted dimensions loop
    size_t loop_size = 1;
    size_t contracted_dims[MAX_ND];
    for (size_t p = 0; p < num_contracted; ++p)
    {
        contracted_dims[p] = d_a_shape[d_contracted_axes_a[p]];
        loop_size *= contracted_dims[p];
    }

    T sum = (T)0.0f;
    for (size_t k = 0; k < loop_size; ++k)
    {
        size_t temp_k = k;

        // Set contracted indices for A and B
        for (size_t p = 0; p < num_contracted; ++p)
        {
            const size_t dim_idx = num_contracted - p - 1;
            const size_t dim_size = contracted_dims[dim_idx];
            const size_t idx = temp_k % dim_size;
            temp_k /= dim_size;

            a_coords[d_contracted_axes_a[dim_idx]] = idx;
            b_coords[d_contracted_axes_b[dim_idx]] = idx;
        }

        // Calculate offsets
        size_t a_offset = 0, b_offset = 0;
        for (size_t d = 0; d < a_nd; ++d)
            a_offset += a_coords[d] * d_a_strides[d];
        for (size_t d = 0; d < b_nd; ++d)
            b_offset += b_coords[d] * d_b_strides[d];

        sum = sum + d_a[a_offset] * d_b[b_offset];
    }

    d_out[out_idx] = sum;
}

// Main tensordot function
template <typename T>
hipError_t tomoTensordot(
    const T *d_a, const T *d_b, T *d_out,
    const size_t *a_shape, size_t a_shape_len,
    const size_t *b_shape, size_t b_shape_len,
    const size_t *out_shape, size_t out_shape_len,
    const size_t *a_strides, size_t a_strides_len,
    const size_t *b_strides, size_t b_strides_len,
    const size_t *out_strides, size_t out_strides_len,
    const size_t *contracted_axes_a, size_t contracted_axes_a_len,
    const size_t *contracted_axes_b, size_t contracted_axes_b_len,
    size_t a_nd, size_t b_nd, size_t out_nd,
    size_t out_size, size_t num_contracted,
    hipStream_t stream)
{
    // Validate inputs
    if (a_shape_len != a_nd || b_shape_len != b_nd || out_shape_len != out_nd ||
        a_strides_len != a_nd || b_strides_len != b_nd || out_strides_len != out_nd ||
        contracted_axes_a_len != num_contracted || contracted_axes_b_len != num_contracted)
    {
        return hipErrorInvalidValue;
    }

    // Device buffers
    size_t *d_a_shape, *d_b_shape, *d_out_shape;
    size_t *d_a_strides, *d_b_strides, *d_out_strides;
    size_t *d_contracted_axes_a, *d_contracted_axes_b;

    CHECK_CUDA(hipMallocAsync(&d_a_shape, a_nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_b_shape, b_nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, out_nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_a_strides, a_nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_b_strides, b_nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_strides, out_nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_contracted_axes_a, num_contracted * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_contracted_axes_b, num_contracted * sizeof(size_t), stream));

    // Copy data to device
    CHECK_CUDA(hipMemcpyAsync(d_a_shape, a_shape, a_nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_b_shape, b_shape, b_nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, out_nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_a_strides, a_strides, a_nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_b_strides, b_strides, b_nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_strides, out_strides, out_nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_contracted_axes_a, contracted_axes_a, num_contracted * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_contracted_axes_b, contracted_axes_b, num_contracted * sizeof(size_t), hipMemcpyHostToDevice, stream));

    // Launch kernel
    const int threads = 256;
    const int blocks = ((int)out_size + threads - 1) / threads;
    tomoTensordotKernel<T><<<blocks, threads, 0, stream>>>(
        d_a, d_b, d_out,
        d_a_shape, d_b_shape, d_out_shape,
        d_a_strides, d_b_strides, d_out_strides,
        a_nd, b_nd, out_nd,
        out_size, num_contracted,
        d_contracted_axes_a, d_contracted_axes_b);

    CHECK_CUDA(hipGetLastError());

    // Free device memory
    CHECK_CUDA(hipFreeAsync(d_a_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_b_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_out_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_a_strides, stream));
    CHECK_CUDA(hipFreeAsync(d_b_strides, stream));
    CHECK_CUDA(hipFreeAsync(d_out_strides, stream));
    CHECK_CUDA(hipFreeAsync(d_contracted_axes_a, stream));
    CHECK_CUDA(hipFreeAsync(d_contracted_axes_b, stream));

    return hipSuccess;
}

template <typename T>
__global__ void tomoTransposeExKernel(
    T const *d_in,
    T *d_out,
    size_t const *d_in_shape,
    size_t const *d_out_shape,
    size_t const *d_in_strides,
    size_t const *d_out_strides, // Included for API consistency, not used
    size_t const *d_perm,
    size_t const nd,
    size_t const out_size)
{
    size_t out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
        return;

    // Unravel out_idx to out_coord
    size_t out_coord[MAX_ND];
    size_t tmp = out_idx;
    for (size_t d = nd; d-- > 0;)
    {
        out_coord[d] = tmp % d_out_shape[d];
        tmp /= d_out_shape[d];
    }

    // Compute input_coord using permutation
    size_t input_coord[MAX_ND];
    for (size_t i = 0; i < nd; ++i)
    {
        size_t p = d_perm[i];
        input_coord[p] = out_coord[i];
    }

    // Compute in_idx using input strides
    size_t in_idx = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        in_idx += input_coord[d] * d_in_strides[d];
    }

    // Copy element
    d_out[out_idx] = d_in[in_idx];
}

template <typename T>
hipError_t tomoTransposeEx(
    T const *d_in,
    T *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *out_shape, size_t const out_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const *out_strides, size_t const out_strides_len,
    size_t const *perm, size_t const perm_len,
    size_t const nd,
    size_t const in_size,
    size_t const out_size,
    hipStream_t const stream)
{
    // Validate dimensions
    if (nd > MAX_ND)
        return hipErrorInvalidValue;
    if (in_shape_len != nd || out_shape_len != nd || in_strides_len != nd ||
        out_strides_len != nd || perm_len != nd)
        return hipErrorInvalidValue;

    // Validate permutation
    bool seen[MAX_ND] = {false};
    for (size_t i = 0; i < nd; ++i)
    {
        size_t p = perm[i];
        if (p >= nd || seen[p])
            return hipErrorInvalidValue;
        seen[p] = true;
    }

    // Validate output shape
    for (size_t i = 0; i < nd; ++i)
    {
        if (out_shape[i] != in_shape[perm[i]])
            return hipErrorInvalidValue;
    }

    // Allocate device memory
    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides, *d_perm;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_perm, nd * sizeof(size_t), stream));

    // Copy data to device
    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_perm, perm, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));

    // Launch kernel
    const int threads = 256;
    const int blocks = ((int)out_size + threads - 1) / threads;
    tomoTransposeExKernel<T><<<blocks, threads, 0, stream>>>(
        d_in, d_out, d_in_shape, d_out_shape, d_in_strides, d_out_strides, d_perm, nd, out_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        return err;

    // Free device memory
    hipFreeAsync(d_in_shape, stream);
    hipFreeAsync(d_out_shape, stream);
    hipFreeAsync(d_in_strides, stream);
    hipFreeAsync(d_out_strides, stream);
    hipFreeAsync(d_perm, stream);

    return hipSuccess;
}

void computeRollaxisPerm(size_t nd, size_t axis, size_t start, size_t *perm)
{
    size_t temp_perm[MAX_ND];
    size_t idx = 0;
    // Collect all axes except the one to roll
    for (size_t i = 0; i < nd; ++i)
    {
        if (i != axis)
        {
            temp_perm[idx++] = i;
        }
    }
    // Adjust start if it exceeds the number of remaining axes
    if (start > idx)
    {
        start = idx;
    }
    // Build the permutation: axes before start, axis, axes after start
    for (size_t i = 0; i < start; ++i)
    {
        perm[i] = temp_perm[i];
    }
    perm[start] = axis;
    for (size_t i = start; i < nd - 1; ++i)
    {
        perm[i + 1] = temp_perm[i];
    }
}

template <typename T>
hipError_t tomoRollaxis(
    T const *d_in,               // Input tensor on device
    T *d_out,                    // Output tensor on device
    size_t const *in_shape,      // Input shape array
    size_t const in_shape_len,   // Length of in_shape
    size_t const *in_strides,    // Input strides array
    size_t const in_strides_len, // Length of in_strides
    size_t const axis,           // Axis to roll
    size_t const start,          // Target position
    size_t const nd,             // Number of dimensions
    size_t const in_size,        // Total input elements
    size_t const out_size,       // Total output elements (should equal in_size)
    hipStream_t const stream    // CUDA stream for async execution
)
{
    // Validate inputs
    if (nd > MAX_ND || in_shape_len != nd || in_strides_len != nd ||
        axis >= nd || start > nd || in_size != out_size)
    {
        return hipErrorInvalidValue;
    }

    // Compute permutation
    size_t perm[MAX_ND];
    computeRollaxisPerm(nd, axis, start, perm);

    // Compute output shape
    size_t out_shape[MAX_ND];
    for (size_t i = 0; i < nd; ++i)
    {
        out_shape[i] = in_shape[perm[i]];
    }

    // Allocate device memory for arrays
    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_perm;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_perm, nd * sizeof(size_t), stream));

    // Copy data to device
    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_perm, perm, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));

    // Launch kernel (assuming tomoTransposeExKernel is defined)
    const int threads = 256;
    const int blocks = (static_cast<int>(out_size) + threads - 1) / threads;
    tomoTransposeExKernel<T><<<blocks, threads, 0, stream>>>(
        d_in, d_out, d_in_shape, d_out_shape, d_in_strides, nullptr, d_perm, nd, out_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        return err;

    // Free device memory
    hipFreeAsync(d_in_shape, stream);
    hipFreeAsync(d_out_shape, stream);
    hipFreeAsync(d_in_strides, stream);
    hipFreeAsync(d_perm, stream);
    return hipSuccess;
}

template <typename T>
__global__ void tomoSwapaxesExKernel(
    T const *d_in, T *d_out,
    size_t const *d_in_shape, size_t const *d_out_shape,
    size_t const *d_in_strides, size_t const *d_out_strides,
    size_t axis1, size_t axis2, size_t nd, size_t out_size)
{
    size_t out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (out_idx >= out_size)
        return;

    // Compute multi-dimensional output coordinates
    size_t out_coord[MAX_ND];
    size_t tmp = out_idx;
    for (size_t d = nd; d-- > 0;)
    {
        out_coord[d] = tmp % d_out_shape[d];
        tmp /= d_out_shape[d];
    }

    // Map to input coordinates by swapping axis1 and axis2
    size_t in_coord[MAX_ND];
    for (size_t d = 0; d < nd; ++d)
    {
        if (d == axis1)
        {
            in_coord[d] = out_coord[axis2];
        }
        else if (d == axis2)
        {
            in_coord[d] = out_coord[axis1];
        }
        else
        {
            in_coord[d] = out_coord[d];
        }
    }

    // Compute input index using strides
    size_t in_idx = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        in_idx += in_coord[d] * d_in_strides[d];
    }

    // Copy element from input to output
    d_out[out_idx] = d_in[in_idx];
}

template <typename T>
hipError_t tomoSwapaxes(
    T const *d_in,              // Input tensor on device
    T *d_out,                   // Output tensor on device
    size_t const *in_shape,     // Input shape array
    size_t in_shape_len,        // Length of in_shape
    size_t const *out_shape,    // Output shape array
    size_t out_shape_len,       // Length of out_shape
    size_t const *in_strides,   // Input strides array
    size_t in_strides_len,      // Length of in_strides
    size_t const *out_strides,  // Output strides array
    size_t out_strides_len,     // Length of out_strides
    size_t axis1, size_t axis2, // Axes to swap
    size_t nd,                  // Number of dimensions
    size_t in_size,             // Total input elements
    size_t out_size,            // Total output elements
    hipStream_t stream)        // CUDA stream for async execution
{
    // Input validation
    if (nd > MAX_ND || in_shape_len != nd || out_shape_len != nd ||
        in_strides_len != nd || out_strides_len != nd ||
        axis1 >= nd || axis2 >= nd || in_size != out_size)
    {
        return hipErrorInvalidValue;
    }

    // Verify output shape matches swapped input shape
    for (size_t d = 0; d < nd; ++d)
    {
        size_t expected = (d == axis1) ? in_shape[axis2] : (d == axis2) ? in_shape[axis1]
                                                                        : in_shape[d];
        if (out_shape[d] != expected)
        {
            return hipErrorInvalidValue;
        }
    }

    // Allocate device memory for shapes and strides
    size_t *d_in_shape, *d_out_shape, *d_in_strides, *d_out_strides;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream));

    // Copy shape and stride data to device
    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t), hipMemcpyHostToDevice, stream));

    // Launch kernel
    const int threads = 256;
    const int blocks = (int)(out_size + threads - 1) / threads;
    tomoSwapaxesExKernel<T><<<blocks, threads, 0, stream>>>(
        d_in, d_out, d_in_shape, d_out_shape, d_in_strides, d_out_strides,
        axis1, axis2, nd, out_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        return err;

    // Free device memory
    hipFreeAsync(d_in_shape, stream);
    hipFreeAsync(d_out_shape, stream);
    hipFreeAsync(d_in_strides, stream);
    hipFreeAsync(d_out_strides, stream);
    return hipSuccess;
}

//------------------------------------------------------------------------------
// tomoIm2colKernel
//------------------------------------------------------------------------------
// Converts an image with shape (n, c, h, w) into a column tensor of shape
// (n, c, kh, kw, out_h, out_w). The convolution parameters are given by kernel
// size (kh, kw), stride (sy, sx), pad (ph, pw) and dilation (dy, dx).
//------------------------------------------------------------------------------
// tomoIm2colKernel (same logic, one thread per element of d_col)
//------------------------------------------------------------------------------
template <typename T>
__global__ void tomoIm2colKernel(
    T const *d_img, T *d_col,
    size_t const n, size_t const c, size_t const h, size_t const w,
    size_t const kh, size_t const kw,
    size_t const out_h, size_t const out_w,
    size_t const sy, size_t const sx,
    size_t const ph, size_t const pw,
    size_t const dy, size_t const dx)
{
    // Each thread processes one element in the "col" space
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = n * c * kh * kw * out_h * out_w;
    if (idx >= total)
        return;

    // Unravel idx into 6 coordinates: [n_idx, c_idx, k_y, k_x, out_y, out_x]
    size_t dims[6] = {n, c, kh, kw, out_h, out_w};
    size_t coords[6];
    {
        size_t tmp = idx;
        for (ptrdiff_t d = 5; d >= 0; --d)
        {
            coords[d] = tmp % dims[d];
            tmp /= dims[d];
        }
    }
    size_t n_idx = coords[0];
    size_t c_idx = coords[1];
    size_t k_y = coords[2];
    size_t k_x = coords[3];
    size_t out_y = coords[4];
    size_t out_x = coords[5];

    // Compute the corresponding input coordinate
    int in_y = static_cast<int>(k_y * dy + out_y * sy) - static_cast<int>(ph);
    int in_x = static_cast<int>(k_x * dx + out_x * sx) - static_cast<int>(pw);

    T value = (T)0;
    if (in_y >= 0 && in_y < static_cast<int>(h) &&
        in_x >= 0 && in_x < static_cast<int>(w))
    {
        // input index
        size_t img_idx = n_idx * (c * h * w) + c_idx * (h * w) + (size_t)in_y * w + (size_t)in_x;
        value = d_img[img_idx];
    }

    // Write to d_col
    d_col[idx] = value;
}

//------------------------------------------------------------------------------
// tomoCol2imKernel (FIXED with atomicAdd or at least one-thread-per-col-element)
//------------------------------------------------------------------------------
// We do: one thread per element in the column tensor, then use atomicAdd on d_img.
template <typename T>
__global__ void tomoCol2imKernel(
    T const *d_col,
    T *d_img,
    size_t const n, size_t const c,
    size_t const h, size_t const w,
    size_t const kh, size_t const kw,
    size_t const out_h, size_t const out_w,
    size_t const sy, size_t const sx,
    size_t const ph, size_t const pw,
    size_t const dx, size_t const dy)
{
    // Each thread processes one element in the "col" space
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = n * c * kh * kw * out_h * out_w;
    if (idx >= total)
        return;

    // Unravel idx into (n, c, kh, kw, out_h, out_w)
    size_t dims[6] = {n, c, kh, kw, out_h, out_w};
    size_t coords[6];
    {
        size_t tmp = idx;
        for (ptrdiff_t d = 5; d >= 0; --d)
        {
            coords[d] = tmp % dims[d];
            tmp /= dims[d];
        }
    }
    size_t n_idx = coords[0];
    size_t c_idx = coords[1];
    size_t k_y = coords[2];
    size_t k_x = coords[3];
    size_t out_y = coords[4];
    size_t out_x = coords[5];

    T val = d_col[idx]; // contribution from this col element

    // Compute the corresponding input coordinate
    int in_y = static_cast<int>(k_y * dy + out_y * sy) - static_cast<int>(ph);
    int in_x = static_cast<int>(k_x * dx + out_x * sx) - static_cast<int>(pw);

    // If inside the image, use atomicAdd to sum partial contributions
    if (in_y >= 0 && in_y < static_cast<int>(h) &&
        in_x >= 0 && in_x < static_cast<int>(w))
    {
        size_t img_idx = n_idx * (c * h * w) + c_idx * (h * w) + (size_t)in_y * w + (size_t)in_x;

        // Use atomicAdd. Implementation differs for half/bfloat16 vs. float/double
        if constexpr (std::is_same_v<T, __half_raw>)
        {
            atomicAdd(reinterpret_cast<__half *>(&d_img[img_idx]),
                      static_cast<__half>(val));
        }
        else if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
        {
            atomicAdd(reinterpret_cast<__hip_bfloat16 *>(&d_img[img_idx]),
                      static_cast<__hip_bfloat16>(val));
        }
        else
        {
            atomicAdd(&d_img[img_idx], val);
        }
    }
}

//------------------------------------------------------------------------------
// Host wrappers with updated usage
//------------------------------------------------------------------------------
template <typename T>
hipError_t tomoIm2col(
    T const *d_img, T *d_col,
    size_t const n, size_t const c,
    size_t const h, size_t const w,
    size_t const kh, size_t const kw,
    size_t const sy, size_t const sx,
    size_t const ph, size_t const pw,
    size_t const dy, size_t const dx,
    hipStream_t stream)
{
    size_t const out_h = (h + 2 * ph - (kh - 1) * dy - 1) / sy + 1; // more general formula
    size_t const out_w = (w + 2 * pw - (kw - 1) * dx - 1) / sx + 1;
    size_t const total = n * c * kh * kw * out_h * out_w;

    int threads = 256;
    int blocks = (int)((total + threads - 1) / threads);

    tomoIm2colKernel<T><<<blocks, threads, 0, stream>>>(
        d_img, d_col,
        n, c, h, w,
        kh, kw, out_h, out_w,
        sy, sx, ph, pw,
        dy, dx);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        return err;

    return hipSuccess;
}

template <typename T>
hipError_t tomoCol2im(
    T const *d_col, T *d_img,
    size_t const n, size_t const c,
    size_t const h, size_t const w,
    size_t const kh, size_t const kw,
    size_t const sy, size_t const sx,
    size_t const ph, size_t const pw,
    size_t const dx, size_t const dy,
    hipStream_t stream)
{
    // We can zero the output first because we'll do atomicAdd:
    size_t const img_total = n * c * h * w;
    hipError_t err = hipMemsetAsync(d_img, 0, img_total * sizeof(T), stream);
    if (err != hipSuccess)
        return err;

    // compute out_h/out_w in the same general form
    size_t const out_h = (h + 2 * ph - (kh - 1) * dy - 1) / sy + 1;
    size_t const out_w = (w + 2 * pw - (kw - 1) * dx - 1) / sx + 1;

    size_t const col_total = n * c * kh * kw * out_h * out_w;
    int threads = 256;
    int blocks = (int)((col_total + threads - 1) / threads);

    tomoCol2imKernel<T><<<blocks, threads, 0, stream>>>(
        d_col, d_img,
        n, c, h, w,
        kh, kw, out_h, out_w,
        sy, sx, ph, pw,
        dx, dy);
    err = hipGetLastError();
    return err;
}

#include <hip/hip_runtime.h>
#include <type_traits> // for std::is_same_v

//------------------------------------------------------------------------------
// 1D im2col kernel
//------------------------------------------------------------------------------
template <typename T>
__global__ void tomoIm2col1dKernel(
    T const *__restrict__ d_in, // [N, C, L]
    T *__restrict__ d_col,      // [N, C, K, outL]
    size_t n,                   // batch size
    size_t c,                   // channels
    size_t l,                   // input length
    size_t k,                   // kernel size
    size_t outL,                // output length
    size_t stride,
    size_t pad,
    size_t dilation)
{
    // linear index into the total of (n * c * k * outL)
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = n * c * k * outL;
    if (idx >= total)
        return;

    // Unravel idx into 4D coords: (n_idx, c_idx, k_idx, out_idx)
    // dims = [n, c, k, outL]
    size_t dims[4] = {n, c, k, outL};
    size_t coords[4];
    {
        size_t tmp = idx;
        for (int d = 3; d >= 0; --d)
        {
            coords[d] = tmp % dims[d];
            tmp /= dims[d];
        }
    }
    size_t n_idx = coords[0];
    size_t c_idx = coords[1];
    size_t k_idx = coords[2];
    size_t out_idx = coords[3];

    // Compute the corresponding input index:
    // in_idx = out_idx*stride - pad + k_idx*dilation
    int in_idx = static_cast<int>(out_idx * stride) - static_cast<int>(pad) + static_cast<int>(k_idx * dilation);

    // read value from input if in range
    T value = T(0);
    if (in_idx >= 0 && in_idx < static_cast<int>(l))
    {
        size_t in_linear = n_idx * (c * l) + c_idx * l + (size_t)in_idx;
        value = d_in[in_linear];
    }

    // write into d_col
    d_col[idx] = value;
}

//------------------------------------------------------------------------------
// 1D col2im kernel
//------------------------------------------------------------------------------
template <typename T>
__global__ void tomoCol2im1dKernel(
    T const *__restrict__ d_col, // [N, C, K, outL]
    T *__restrict__ d_in,        // [N, C, L]
    size_t n,
    size_t c,
    size_t l,
    size_t k,
    size_t outL,
    size_t stride,
    size_t pad,
    size_t dilation)
{
    // linear index into [N, C, K, outL]
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = n * c * k * outL;
    if (idx >= total)
        return;

    // unravel idx -> (n_idx, c_idx, k_idx, out_idx)
    size_t dims[4] = {n, c, k, outL};
    size_t coords[4];
    {
        size_t tmp = idx;
        for (int d = 3; d >= 0; --d)
        {
            coords[d] = tmp % dims[d];
            tmp /= dims[d];
        }
    }
    size_t n_idx = coords[0];
    size_t c_idx = coords[1];
    size_t k_idx = coords[2];
    size_t out_idx = coords[3];

    T val = d_col[idx];

    // compute the input index
    int in_idx = static_cast<int>(out_idx * stride) - static_cast<int>(pad) + static_cast<int>(k_idx * dilation);

    if (in_idx >= 0 && in_idx < static_cast<int>(l))
    {
        size_t in_linear = n_idx * (c * l) + c_idx * l + (size_t)in_idx;

        // If you also want half/bfloat16 support with atomicAdd:
        //   if constexpr (std::is_same_v<T, __half_raw>) { ... }
        //   else if constexpr (std::is_same_v<T, __hip_bfloat16_raw>) { ... }
        //   else { atomicAdd(...); }

        if constexpr (std::is_same_v<T, __half_raw>)
        {
            atomicAdd(reinterpret_cast<__half *>(&d_in[in_linear]),
                      static_cast<__half>(val));
        }
        else if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
        {
            atomicAdd(reinterpret_cast<__hip_bfloat16 *>(&d_in[in_linear]),
                      static_cast<__hip_bfloat16>(val));
        }
        else
        {
            atomicAdd(&d_in[in_linear], val);
        }
    }
}

template <typename T>
hipError_t tomoIm2col1d(
    T const *d_in,
    T *d_col,
    size_t n, // batch
    size_t c, // channels
    size_t l, // input length
    size_t k, // kernel size
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    // compute outL
    //   outL = floor( (l + 2*pad - (k-1)*dilation - 1) / stride ) + 1
    size_t outL = (l + 2 * pad - (k - 1) * dilation - 1) / stride + 1;

    // total threads = n*c*k*outL
    size_t total = n * c * k * outL;
    int threads = 256;
    int blocks = (int)((total + threads - 1) / threads);

    tomoIm2col1dKernel<T><<<blocks, threads, 0, stream>>>(
        d_in, d_col,
        n, c, l, k, outL,
        stride, pad, dilation);

    return hipGetLastError();
}

template <typename T>
hipError_t tomoCol2im1d(
    T const *d_col,
    T *d_in,
    size_t n, // batch
    size_t c, // channels
    size_t l, // input length
    size_t k, // kernel size
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    // Optionally zero out d_in first if you want to sum partial results:
    hipError_t err = hipMemsetAsync(d_in, 0, n * c * l * sizeof(T), stream);
    if (err != hipSuccess)
        return err;

    // compute outL same as above
    size_t outL = (l + 2 * pad - (k - 1) * dilation - 1) / stride + 1;

    size_t total = n * c * k * outL;
    int threads = 256;
    int blocks = (int)((total + threads - 1) / threads);

    tomoCol2im1dKernel<T><<<blocks, threads, 0, stream>>>(
        d_col, d_in,
        n, c, l, k, outL,
        stride, pad, dilation);

    return hipGetLastError();
}

// Forward declaration so we can partially specialize.
template <typename T>
__device__ inline T deviceInfinity(bool negative);

// Specialize for float
template <>
__device__ inline float deviceInfinity<float>(bool negative)
{
    return negative ? -std::numeric_limits<float>::infinity()
                    : std::numeric_limits<float>::infinity();
}

// Specialize for double
template <>
__device__ inline double deviceInfinity<double>(bool negative)
{
    return negative ? -std::numeric_limits<double>::infinity()
                    : std::numeric_limits<double>::infinity();
}

// Specialize for __half_raw
// We do a simple conversion from float∞ to half∞.
template <>
__device__ inline __half_raw deviceInfinity<__half_raw>(bool negative)
{
    float inf = negative ? -std::numeric_limits<float>::infinity()
                         : std::numeric_limits<float>::infinity();
    // This cast depends on your environment; you may need __float2half_rn(inf).
    // For “raw” half, do a reinterpret if you already have an operator.
    // If no operator is available, define a custom conversion.
    // For demonstration, assume direct C-style cast is valid:
    return (__half_raw)inf;
}

// Specialize for __hip_bfloat16_raw
template <>
__device__ inline __hip_bfloat16_raw deviceInfinity<__hip_bfloat16_raw>(bool negative)
{
    float inf = negative ? -std::numeric_limits<float>::infinity()
                         : std::numeric_limits<float>::infinity();
    // Similarly for bfloat16.
    return (__hip_bfloat16_raw)inf;
}

template <typename T>
__global__ void tomoArgmaxKernel(
    const T *d_in, // Input data
    const size_t *in_shape,
    const size_t *in_strides,
    size_t *d_out, // Output indices
    const size_t *out_shape,
    const size_t *out_strides,
    size_t out_size,
    size_t nd)
{
    // Each block handles one output element
    size_t out_idx = blockIdx.x;
    if (out_idx >= out_size)
        return;

    // Compute output coords from out_idx
    size_t out_coords[MAX_ND];
    {
        size_t tmp = out_idx;
        for (ptrdiff_t d = nd - 1; d >= 0; --d)
        {
            out_coords[d] = tmp % out_shape[d];
            tmp /= out_shape[d];
        }
    }

    // Compute output offset
    size_t out_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        out_offset += out_coords[d] * out_strides[d];
    }

    // Compute base offset in the input
    size_t base_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        size_t in_c = (out_shape[d] == 1) ? 0 : out_coords[d];
        base_offset += in_c * in_strides[d];
    }

    // Identify reduced dimensions
    size_t reduced_dims[MAX_ND];
    size_t reduced_sizes[MAX_ND];
    size_t num_reduced = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        // If out_shape[d] == 1 but in_shape[d] > 1 => dimension is reduced
        if (out_shape[d] == 1 && in_shape[d] > 1)
        {
            reduced_dims[num_reduced] = d;
            reduced_sizes[num_reduced] = in_shape[d];
            num_reduced++;
        }
    }

    // Total # of elements in reduced dimensions
    size_t N = 1;
    for (size_t j = 0; j < num_reduced; ++j)
    {
        N *= reduced_sizes[j];
    }

    // Each thread finds a local maximum over its subset
    T local_max = deviceInfinity<T>(/*negative=*/true); // negative infinity
    size_t local_argmax = 0;

    for (size_t i = threadIdx.x; i < N; i += blockDim.x)
    {
        // Unravel i into coords along the reduced dims
        size_t reduced_coords[MAX_ND];
        {
            size_t tmp_i = i;
            for (ptrdiff_t j = num_reduced - 1; j >= 0; --j)
            {
                reduced_coords[j] = tmp_i % reduced_sizes[j];
                tmp_i /= reduced_sizes[j];
            }
        }

        // Compute input offset
        size_t offset = base_offset;
        for (size_t j = 0; j < num_reduced; ++j)
        {
            size_t d = reduced_dims[j];
            offset += reduced_coords[j] * in_strides[d];
        }

        // Update local max and argmax
        T val = d_in[offset];
        if (val > local_max)
        {
            local_max = val;
            local_argmax = i;
        }
    }

    // Use shared memory for block-level reduction
    extern __shared__ char shared_mem[];
    // We place an array of T, then an array of size_t
    T *shared_max = reinterpret_cast<T *>(shared_mem);
    size_t *shared_argmax = reinterpret_cast<size_t *>(shared_max + blockDim.x);

    shared_max[threadIdx.x] = local_max;
    shared_argmax[threadIdx.x] = local_argmax;
    __syncthreads();

    // Parallel reduction by half
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (shared_max[threadIdx.x + s] > shared_max[threadIdx.x])
            {
                shared_max[threadIdx.x] = shared_max[threadIdx.x + s];
                shared_argmax[threadIdx.x] = shared_argmax[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // Thread 0 writes out final argmax
    if (threadIdx.x == 0)
    {
        d_out[out_offset] = shared_argmax[0];
    }
}

template <typename T>
hipError_t tomoArgmax(
    const T *d_in,
    size_t *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    // Validate shape/stride array lengths
    if (in_shape_len != nd || out_shape_len != nd ||
        in_strides_len != nd || out_strides_len != nd)
    {
        return hipErrorInvalidValue;
    }

    // Allocate device buffers for shapes & strides
    size_t *d_in_shape, *d_out_shape;
    size_t *d_in_strides, *d_out_strides;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream));

    // Copy to device
    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));

    // Launch kernel
    const int threads = 256; // must be power-of-two for that simple reduction
    const int blocks = static_cast<int>(out_size);
    // shared memory:
    //  each thread has a T plus a size_t
    const int smem_sz = threads * (int(sizeof(T)) + int(sizeof(size_t)));

    tomoArgmaxKernel<T><<<blocks, threads, smem_sz, stream>>>(
        d_in, d_in_shape, d_in_strides,
        d_out, d_out_shape, d_out_strides,
        out_size, nd);
    CHECK_CUDA(hipGetLastError());

    // Cleanup
    CHECK_CUDA(hipFreeAsync(d_in_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_out_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_in_strides, stream));
    CHECK_CUDA(hipFreeAsync(d_out_strides, stream));

    return hipSuccess;
}

template <typename T>
__global__ void tomoArgminKernel(
    const T *d_in,
    const size_t *in_shape,
    const size_t *in_strides,
    size_t *d_out,
    const size_t *out_shape,
    const size_t *out_strides,
    size_t out_size,
    size_t nd)
{
    size_t out_idx = blockIdx.x;
    if (out_idx >= out_size)
        return;

    // Compute output coords
    size_t out_coords[MAX_ND];
    {
        size_t tmp = out_idx;
        for (ptrdiff_t d = nd - 1; d >= 0; --d)
        {
            out_coords[d] = tmp % out_shape[d];
            tmp /= out_shape[d];
        }
    }

    // Output offset
    size_t out_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        out_offset += out_coords[d] * out_strides[d];
    }

    // Base offset in input
    size_t base_offset = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        size_t in_c = (out_shape[d] == 1) ? 0 : out_coords[d];
        base_offset += in_c * in_strides[d];
    }

    // Reduced dimensions
    size_t reduced_dims[MAX_ND];
    size_t reduced_sizes[MAX_ND];
    size_t num_reduced = 0;
    for (size_t d = 0; d < nd; ++d)
    {
        if (out_shape[d] == 1 && in_shape[d] > 1)
        {
            reduced_dims[num_reduced] = d;
            reduced_sizes[num_reduced] = in_shape[d];
            num_reduced++;
        }
    }

    size_t N = 1;
    for (size_t j = 0; j < num_reduced; ++j)
    {
        N *= reduced_sizes[j];
    }

    // Initialize local min to +∞
    T local_min = deviceInfinity<T>(/*negative=*/false);
    size_t local_argmin = 0;

    for (size_t i = threadIdx.x; i < N; i += blockDim.x)
    {
        // Unravel i -> reduced_coords
        size_t reduced_coords[MAX_ND];
        {
            size_t tmp_i = i;
            for (ptrdiff_t j = num_reduced - 1; j >= 0; --j)
            {
                reduced_coords[j] = tmp_i % reduced_sizes[j];
                tmp_i /= reduced_sizes[j];
            }
        }

        // Input offset
        size_t offset = base_offset;
        for (size_t j = 0; j < num_reduced; ++j)
        {
            size_t d = reduced_dims[j];
            offset += reduced_coords[j] * in_strides[d];
        }

        // Compare & update local min
        T val = d_in[offset];
        if (val < local_min)
        {
            local_min = val;
            local_argmin = i;
        }
    }

    // Shared memory reduction
    extern __shared__ char shared_mem[];
    T *shared_min = reinterpret_cast<T *>(shared_mem);
    size_t *shared_argmin = reinterpret_cast<size_t *>(shared_min + blockDim.x);

    shared_min[threadIdx.x] = local_min;
    shared_argmin[threadIdx.x] = local_argmin;
    __syncthreads();

    for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            if (shared_min[threadIdx.x + s] < shared_min[threadIdx.x])
            {
                shared_min[threadIdx.x] = shared_min[threadIdx.x + s];
                shared_argmin[threadIdx.x] = shared_argmin[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // Write final argmin
    if (threadIdx.x == 0)
    {
        d_out[out_offset] = shared_argmin[0];
    }
}

template <typename T>
hipError_t tomoArgmin(
    const T *d_in,
    size_t *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    if (in_shape_len != nd || out_shape_len != nd ||
        in_strides_len != nd || out_strides_len != nd)
    {
        return hipErrorInvalidValue;
    }

    size_t *d_in_shape, *d_out_shape;
    size_t *d_in_strides, *d_out_strides;
    CHECK_CUDA(hipMallocAsync(&d_in_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_shape, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_in_strides, nd * sizeof(size_t), stream));
    CHECK_CUDA(hipMallocAsync(&d_out_strides, nd * sizeof(size_t), stream));

    CHECK_CUDA(hipMemcpyAsync(d_in_shape, in_shape, nd * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_shape, out_shape, nd * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_in_strides, in_strides, nd * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(d_out_strides, out_strides, nd * sizeof(size_t),
                               hipMemcpyHostToDevice, stream));

    const int threads = 256;
    const int blocks = static_cast<int>(out_size);
    const int smem_sz = threads * (int(sizeof(T)) + int(sizeof(size_t)));

    tomoArgminKernel<T><<<blocks, threads, smem_sz, stream>>>(
        d_in, d_in_shape, d_in_strides,
        d_out, d_out_shape, d_out_strides,
        out_size, nd);
    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipFreeAsync(d_in_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_out_shape, stream));
    CHECK_CUDA(hipFreeAsync(d_in_strides, stream));
    CHECK_CUDA(hipFreeAsync(d_out_strides, stream));
    return hipSuccess;
}

template <typename T>
__global__ void tomoMaxPool2dForwardKernel(
    const T *__restrict__ input, // [N, C, H, W]
    T *__restrict__ output,      // [N, C, outH, outW]
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW)
{
    // Each thread corresponds to one element in output: [N, C, outH, outW]
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = N * C * outH * outW;
    if (idx >= total)
        return;

    // unravel idx -> (n, c, oh, ow)
    size_t ow = idx % outW;
    size_t tmp = idx / outW;
    size_t oh = tmp % outH;
    tmp /= outH;
    size_t c = tmp % C;
    size_t n = tmp / C;

    // compute the “start/end” in input image
    size_t in_start_h = oh * strideH - padH;
    size_t in_start_w = ow * strideW - padW;
    size_t in_end_h = in_start_h + kernelH;
    size_t in_end_w = in_start_w + kernelW;

    // T max_val = static_cast<T>(-FLT_MAX); // or -FLT_MAX for float
    T max_val = deviceInfinity<T>(true); // or -FLT_MAX for float
    for (size_t ih = in_start_h; ih < in_end_h; ih++)
    {
        for (size_t iw = in_start_w; iw < in_end_w; iw++)
        {
            if (ih < H && iw < W)
            {
                size_t in_index = (n * C + c) * (H * W) + ih * W + iw;
                T val = input[in_index];
                if (val > max_val)
                {
                    max_val = val;
                }
            }
        }
    }
    output[idx] = max_val;
}

template <typename T>
hipError_t tomoMaxPool2dForward(
    const T *input,
    T *output,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW,
    hipStream_t stream)
{
    size_t total = N * C * outH * outW;
    int threads = 256;
    int blocks = (int)((total + threads - 1) / threads);

    tomoMaxPool2dForwardKernel<T><<<blocks, threads, 0, stream>>>(
        input, output,
        N, C, H, W,
        outH, outW,
        kernelH, kernelW,
        strideH, strideW,
        padH, padW);

    return hipGetLastError();
}

template <typename T>
__global__ void tomoMaxPool2dBackwardKernel(
    const T *__restrict__ input,   // [N, C, H, W] (to find max indexes)
    const T *__restrict__ gradOut, // [N, C, outH, outW]
    T *gradIn,                     // [N, C, H, W] (to accumulate)
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW)
{
    // one thread per (n,c,oh,ow)
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = N * C * outH * outW;
    if (idx >= total)
        return;

    size_t ow = idx % outW;
    size_t tmp = idx / outW;
    size_t oh = tmp % outH;
    tmp /= outH;
    size_t c = tmp % C;
    size_t n = tmp / C;

    // figure out region in input
    size_t in_start_h = oh * strideH - padH;
    size_t in_start_w = ow * strideW - padW;
    size_t in_end_h = in_start_h + kernelH;
    size_t in_end_w = in_start_w + kernelW;

    // find which (ih, iw) had the max
    T max_val = deviceInfinity<T>(true);
    ptrdiff_t max_h = -1, max_w = -1;
    for (size_t ih = in_start_h; ih < in_end_h; ih++)
    {
        for (size_t iw = in_start_w; iw < in_end_w; iw++)
        {
            if (ih < H && iw < W)
            {
                size_t in_index = (n * C + c) * (H * W) + ih * W + iw;
                T val = input[in_index];
                if (val > max_val)
                {
                    max_val = val;
                    max_h = (ptrdiff_t)ih;
                    max_w = (ptrdiff_t)iw;
                }
            }
        }
    }

    // Add the upstream gradient to that max location
    T grad_val = gradOut[idx];
    if (max_h >= 0 && max_w >= 0)
    {
        size_t in_index = (n * C + c) * (H * W) + max_h * W + max_w;

        if constexpr (std::is_same_v<T, __half_raw>)
        {
            atomicAdd(reinterpret_cast<__half *>(&gradIn[in_index]),
                      static_cast<__half>(grad_val));
        }
        else if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
        {
            atomicAdd(reinterpret_cast<__hip_bfloat16 *>(&gradIn[in_index]),
                      static_cast<__hip_bfloat16>(grad_val));
        }
        else
        {
            atomicAdd(&gradIn[in_index], grad_val);
        }
    }
}

template <typename T>
hipError_t tomoMaxPool2dBackward(
    const T *input,
    const T *gradOut,
    T *gradIn,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW,
    hipStream_t stream)
{
    // zero gradIn first
    hipError_t err = hipMemsetAsync(gradIn, 0, N * C * H * W * sizeof(T), stream);
    if (err != hipSuccess)
        return err;

    size_t total = (size_t)N * C * outH * outW;
    int threads = 256;
    int blocks = (int)((total + threads - 1) / threads);

    tomoMaxPool2dBackwardKernel<T><<<blocks, threads, 0, stream>>>(
        input, gradOut, gradIn,
        N, C, H, W,
        outH, outW,
        kernelH, kernelW,
        strideH, strideW,
        padH, padW);

    return hipGetLastError();
}

template <typename T>
__global__ void tomoAvgPool2dForwardKernel(
    const T *__restrict__ input, // [N, C, H, W]
    T *__restrict__ output,      // [N, C, outH, outW]
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = N * C * outH * outW;
    if (idx >= total)
        return;

    size_t ow = idx % outW;
    size_t tmp = idx / outW;
    size_t oh = tmp % outH;
    tmp /= outH;
    size_t c = tmp % C;
    size_t n = tmp / C;

    size_t in_start_h = oh * strideH - padH;
    size_t in_start_w = ow * strideW - padW;
    size_t in_end_h = in_start_h + kernelH;
    size_t in_end_w = in_start_w + kernelW;

    T sum_val = static_cast<T>(0.0);
    int count = 0;
    for (size_t ih = in_start_h; ih < in_end_h; ih++)
    {
        for (size_t iw = in_start_w; iw < in_end_w; iw++)
        {
            if (ih < H && iw < W)
            {
                size_t in_index = (n * C + c) * (H * W) + ih * W + iw;
                sum_val = sum_val + input[in_index];
                count++;
            }
        }
    }
    if (count > 0)
    {
        sum_val = sum_val / static_cast<T>(count);
    }
    output[idx] = sum_val;
}

template <typename T>
hipError_t tomoAvgPool2dForward(
    const T *input,
    T *output,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW,
    hipStream_t stream)
{
    size_t total = (size_t)N * C * outH * outW;
    int threads = 256;
    int blocks = (int)((total + threads - 1) / threads);

    tomoAvgPool2dForwardKernel<T><<<blocks, threads, 0, stream>>>(
        input, output,
        N, C, H, W,
        outH, outW,
        kernelH, kernelW,
        strideH, strideW,
        padH, padW);

    return hipGetLastError();
}

template <typename T>
__global__ void tomoAvgPool2dBackwardKernel(
    const T *__restrict__ gradOut, // [N, C, outH, outW]
    T *gradIn,                     // [N, C, H, W]
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = N * C * outH * outW;
    if (idx >= total)
        return;

    size_t ow = idx % outW;
    size_t tmp = idx / outW;
    size_t oh = tmp % outH;
    tmp /= outH;
    int c = tmp % C;
    int n = tmp / C;

    T gradVal = gradOut[idx];

    // figure out region in input
    size_t in_start_h = oh * strideH - padH;
    size_t in_start_w = ow * strideW - padW;
    size_t in_end_h = in_start_h + kernelH;
    size_t in_end_w = in_start_w + kernelW;

    // find how many valid elements are in that region
    size_t count = 0;
    for (size_t ih = in_start_h; ih < in_end_h; ih++)
    {
        for (size_t iw = in_start_w; iw < in_end_w; iw++)
        {
            if (ih < H && iw < W)
            {
                count++;
            }
        }
    }

    if (count == 0)
        return;
    T factor = gradVal / static_cast<T>(count);

    // distribute
    for (size_t ih = in_start_h; ih < in_end_h; ih++)
    {
        for (size_t iw = in_start_w; iw < in_end_w; iw++)
        {
            if (ih < H && iw < W)
            {
                size_t in_index = (n * C + c) * (H * W) + ih * W + iw;
                // atomicAdd(&gradIn[in_index], factor);

                if constexpr (std::is_same_v<T, __half_raw>)
                {
                    atomicAdd(reinterpret_cast<__half *>(&gradIn[in_index]),
                              static_cast<__half>(factor));
                }
                else if constexpr (std::is_same_v<T, __hip_bfloat16_raw>)
                {
                    atomicAdd(reinterpret_cast<__hip_bfloat16 *>(&gradIn[in_index]),
                              static_cast<__hip_bfloat16>(factor));
                }
                else
                {
                    atomicAdd(&gradIn[in_index], factor);
                }
            }
        }
    }
}

template <typename T>
hipError_t tomoAvgPool2dBackward(
    const T *gradOut,
    T *gradIn,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW,
    hipStream_t stream)
{
    // zero gradIn
    hipError_t err = hipMemsetAsync(gradIn, 0, (size_t)N * C * H * W * sizeof(T), stream);
    if (err != hipSuccess)
        return err;

    size_t total = N * C * outH * outW;
    int threads = 256;
    int blocks = (int)((total + threads - 1) / threads);

    tomoAvgPool2dBackwardKernel<T><<<blocks, threads, 0, stream>>>(
        gradOut, gradIn,
        N, C, H, W,
        outH, outW,
        kernelH, kernelW,
        strideH, strideW,
        padH, padW);

    return hipGetLastError();
}

////////////////////////////////////////////////////////////////////////////////
// BROADCAST-TO WRAPPERS
////////////////////////////////////////////////////////////////////////////////

/*
   Half version
   - We assume your half type is __half_raw as in your snippet.
   - If you have a different type or include <hip/hip_fp16.h>, adapt accordingly.
*/
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoBroadcastToH(
    __half_raw const *d_in,
    __half_raw *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoBroadcastTo<__half_raw>(
        d_in,
        d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

/*
   Bfloat16 version
   - We assume your bfloat16 type is __hip_bfloat16_raw as in your snippet.
*/
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoBroadcastToB(
    __hip_bfloat16_raw const *d_in,
    __hip_bfloat16_raw *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoBroadcastTo<__hip_bfloat16_raw>(
        d_in,
        d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

/*
   Float version
*/
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoBroadcastToF(
    float const *d_in,
    float *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoBroadcastTo<float>(
        d_in,
        d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

/*
   Double version
*/
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoBroadcastToD(
    double const *d_in,
    double *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoBroadcastTo<double>(
        d_in,
        d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

////////////////////////////////////////////////////////////////////////////////
// SUM-TO WRAPPERS
////////////////////////////////////////////////////////////////////////////////

// Half
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSumToH(
    __half_raw const *d_in,
    __half_raw *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoSumTo<__half_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSumToB(
    __hip_bfloat16_raw const *d_in,
    __hip_bfloat16_raw *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoSumTo<__hip_bfloat16_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSumToF(
    float const *d_in,
    float *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoSumTo<float>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSumToD(
    double const *d_in,
    double *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoSumTo<double>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoLinearH(
    __half_raw const *A, __half_raw const *B, size_t M, size_t K, size_t N, __half_raw const *bias, __half_raw *C,
    hipStream_t stream)
{
    return tomoLinear<__half_raw>(
        A, B,
        M, K,
        N, bias,
        C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoLinearB(
    __hip_bfloat16_raw const *A, __hip_bfloat16_raw const *B, size_t M, size_t K, size_t N, __hip_bfloat16_raw const *bias, __hip_bfloat16_raw *C,
    hipStream_t stream)
{
    return tomoLinear<__hip_bfloat16_raw>(
        A, B,
        M, K,
        N, bias,
        C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoLinearF(
    float const *A, float const *B, size_t M, size_t K, size_t N, float const *bias, float *C,
    hipStream_t stream)
{
    return tomoLinear<float>(
        A, B,
        M, K,
        N, bias,
        C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoLinearD(
    double const *A, double const *B, size_t M, size_t K, size_t N, double const *bias, double *C,
    hipStream_t stream)
{
    return tomoLinear<double>(
        A, B,
        M, K,
        N, bias,
        C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoLinearImpH(
    __half_raw const *A, __half_raw const *B, size_t M, size_t K, size_t N,
    __half_raw const *bias, __half_raw *C, hipStream_t stream)
{
    return tomoLinearImp<__half_raw>(A, B, M, K, N, bias, C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoLinearImpB(
    __hip_bfloat16_raw const *A, __hip_bfloat16_raw const *B, size_t M, size_t K, size_t N,
    __hip_bfloat16_raw const *bias, __hip_bfloat16_raw *C, hipStream_t stream)
{
    return tomoLinearImp<__hip_bfloat16_raw>(A, B, M, K, N, bias, C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoLinearImpF(
    float const *A, float const *B, size_t M, size_t K, size_t N,
    float const *bias, float *C, hipStream_t stream)
{
    return tomoLinearImp<float>(A, B, M, K, N, bias, C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoLinearImpD(
    double const *A, double const *B, size_t M, size_t K, size_t N,
    double const *bias, double *C, hipStream_t stream)
{
    return tomoLinearImp<double>(A, B, M, K, N, bias, C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTransposeH(__half_raw const *A, size_t M, size_t N, __half_raw *C, hipStream_t stream)
{
    return tomoTranspose<__half_raw>(
        A,
        M,
        N,
        C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTransposeB(__hip_bfloat16_raw const *A, size_t M, size_t N, __hip_bfloat16_raw *C, hipStream_t stream)
{
    return tomoTranspose<__hip_bfloat16_raw>(
        A,
        M,
        N,
        C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTransposeF(float const *A, size_t M, size_t N, float *C, hipStream_t stream)
{
    return tomoTranspose<float>(
        A,
        M,
        N,
        C, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTransposeD(double const *A, size_t M, size_t N, double *C, hipStream_t stream)
{
    return tomoTranspose<double>(
        A,
        M,
        N,
        C, stream);
}

// ----- Max wrappers -----

// Half
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxToH(
    __half_raw const *d_in,
    __half_raw *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoMaxTo<__half_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxToB(
    __hip_bfloat16_raw const *d_in,
    __hip_bfloat16_raw *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoMaxTo<__hip_bfloat16_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxToF(
    float const *d_in,
    float *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoMaxTo<float>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxToD(
    double const *d_in,
    double *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoMaxTo<double>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// ----- Min wrappers -----

// Half
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMinToH(
    __half_raw const *d_in,
    __half_raw *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoMinTo<__half_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMinToB(
    __hip_bfloat16_raw const *d_in,
    __hip_bfloat16_raw *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoMinTo<__hip_bfloat16_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMinToF(
    float const *d_in,
    float *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoMinTo<float>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMinToD(
    double const *d_in,
    double *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t in_size,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoMinTo<double>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        in_size,
        out_size,
        nd,
        stream);
}

// Half (FP16)
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTensordotH(
    __half_raw const *d_a,
    __half_raw const *d_b,
    __half_raw *d_out,
    size_t const *a_shape, size_t a_shape_len,
    size_t const *b_shape, size_t b_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *a_strides, size_t a_strides_len,
    size_t const *b_strides, size_t b_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *contracted_axes_a, size_t contracted_axes_a_len,
    size_t const *contracted_axes_b, size_t contracted_axes_b_len,
    size_t a_nd, size_t b_nd, size_t out_nd,
    size_t out_size, size_t num_contracted,
    hipStream_t stream)
{
    return tomoTensordot<__half_raw>(
        d_a, d_b, d_out,
        a_shape, a_shape_len,
        b_shape, b_shape_len,
        out_shape, out_shape_len,
        a_strides, a_strides_len,
        b_strides, b_strides_len,
        out_strides, out_strides_len,
        contracted_axes_a, contracted_axes_a_len,
        contracted_axes_b, contracted_axes_b_len,
        a_nd, b_nd, out_nd,
        out_size, num_contracted,
        stream);
}

// Bfloat16
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTensordotB(
    __hip_bfloat16_raw const *d_a,
    __hip_bfloat16_raw const *d_b,
    __hip_bfloat16_raw *d_out,
    size_t const *a_shape, size_t a_shape_len,
    size_t const *b_shape, size_t b_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *a_strides, size_t a_strides_len,
    size_t const *b_strides, size_t b_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *contracted_axes_a, size_t contracted_axes_a_len,
    size_t const *contracted_axes_b, size_t contracted_axes_b_len,
    size_t a_nd, size_t b_nd, size_t out_nd,
    size_t out_size, size_t num_contracted,
    hipStream_t stream)
{
    return tomoTensordot<__hip_bfloat16_raw>(
        d_a, d_b, d_out,
        a_shape, a_shape_len,
        b_shape, b_shape_len,
        out_shape, out_shape_len,
        a_strides, a_strides_len,
        b_strides, b_strides_len,
        out_strides, out_strides_len,
        contracted_axes_a, contracted_axes_a_len,
        contracted_axes_b, contracted_axes_b_len,
        a_nd, b_nd, out_nd,
        out_size, num_contracted,
        stream);
}

// Float
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTensordotF(
    float const *d_a,
    float const *d_b,
    float *d_out,
    size_t const *a_shape, size_t a_shape_len,
    size_t const *b_shape, size_t b_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *a_strides, size_t a_strides_len,
    size_t const *b_strides, size_t b_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *contracted_axes_a, size_t contracted_axes_a_len,
    size_t const *contracted_axes_b, size_t contracted_axes_b_len,
    size_t a_nd, size_t b_nd, size_t out_nd,
    size_t out_size, size_t num_contracted,
    hipStream_t stream)
{
    return tomoTensordot<float>(
        d_a, d_b, d_out,
        a_shape, a_shape_len,
        b_shape, b_shape_len,
        out_shape, out_shape_len,
        a_strides, a_strides_len,
        b_strides, b_strides_len,
        out_strides, out_strides_len,
        contracted_axes_a, contracted_axes_a_len,
        contracted_axes_b, contracted_axes_b_len,
        a_nd, b_nd, out_nd,
        out_size, num_contracted,
        stream);
}

// Double
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTensordotD(
    double const *d_a,
    double const *d_b,
    double *d_out,
    size_t const *a_shape, size_t a_shape_len,
    size_t const *b_shape, size_t b_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *a_strides, size_t a_strides_len,
    size_t const *b_strides, size_t b_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t const *contracted_axes_a, size_t contracted_axes_a_len,
    size_t const *contracted_axes_b, size_t contracted_axes_b_len,
    size_t a_nd, size_t b_nd, size_t out_nd,
    size_t out_size, size_t num_contracted,
    hipStream_t stream)
{
    return tomoTensordot<double>(
        d_a, d_b, d_out,
        a_shape, a_shape_len,
        b_shape, b_shape_len,
        out_shape, out_shape_len,
        a_strides, a_strides_len,
        b_strides, b_strides_len,
        out_strides, out_strides_len,
        contracted_axes_a, contracted_axes_a_len,
        contracted_axes_b, contracted_axes_b_len,
        a_nd, b_nd, out_nd,
        out_size, num_contracted,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTransposeExH(
    __half_raw const *d_in, __half_raw *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *out_shape, size_t const out_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const *out_strides, size_t const out_strides_len,
    size_t const *perm, size_t const perm_len,
    size_t const nd,
    size_t const in_size, size_t const out_size,
    hipStream_t const stream)
{
    return tomoTransposeEx<__half_raw>(
        d_in, d_out, in_shape, in_shape_len, out_shape, out_shape_len,
        in_strides, in_strides_len, out_strides, out_strides_len,
        perm, perm_len, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTransposeExB(
    __hip_bfloat16_raw const *d_in, __hip_bfloat16_raw *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *out_shape, size_t const out_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const *out_strides, size_t const out_strides_len,
    size_t const *perm, size_t const perm_len,
    size_t const nd,
    size_t const in_size, size_t const out_size,
    hipStream_t const stream)
{
    return tomoTransposeEx<__hip_bfloat16_raw>(
        d_in, d_out, in_shape, in_shape_len, out_shape, out_shape_len,
        in_strides, in_strides_len, out_strides, out_strides_len,
        perm, perm_len, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTransposeExF(
    float const *d_in, float *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *out_shape, size_t const out_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const *out_strides, size_t const out_strides_len,
    size_t const *perm, size_t const perm_len,
    size_t const nd,
    size_t const in_size, size_t const out_size,
    hipStream_t const stream)
{
    return tomoTransposeEx<float>(
        d_in, d_out, in_shape, in_shape_len, out_shape, out_shape_len,
        in_strides, in_strides_len, out_strides, out_strides_len,
        perm, perm_len, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoTransposeExD(
    double const *d_in, double *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *out_shape, size_t const out_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const *out_strides, size_t const out_strides_len,
    size_t const *perm, size_t const perm_len,
    size_t const nd,
    size_t const in_size, size_t const out_size,
    hipStream_t const stream)
{
    return tomoTransposeEx<double>(
        d_in, d_out, in_shape, in_shape_len, out_shape, out_shape_len,
        in_strides, in_strides_len, out_strides, out_strides_len,
        perm, perm_len, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoRollaxisH(
    __half_raw const *d_in, __half_raw *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const axis, size_t const start,
    size_t const nd, size_t const in_size, size_t const out_size,
    hipStream_t const stream)
{
    return tomoRollaxis<__half_raw>(
        d_in, d_out, in_shape, in_shape_len, in_strides, in_strides_len,
        axis, start, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoRollaxisB(
    __hip_bfloat16_raw const *d_in, __hip_bfloat16_raw *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const axis, size_t const start,
    size_t const nd, size_t const in_size, size_t const out_size,
    hipStream_t const stream)
{
    return tomoRollaxis<__hip_bfloat16_raw>(
        d_in, d_out, in_shape, in_shape_len, in_strides, in_strides_len,
        axis, start, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoRollaxisF(
    float const *d_in, float *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const axis, size_t const start,
    size_t const nd, size_t const in_size, size_t const out_size,
    hipStream_t const stream)
{
    return tomoRollaxis<float>(
        d_in, d_out, in_shape, in_shape_len, in_strides, in_strides_len,
        axis, start, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoRollaxisD(
    double const *d_in, double *d_out,
    size_t const *in_shape, size_t const in_shape_len,
    size_t const *in_strides, size_t const in_strides_len,
    size_t const axis, size_t const start,
    size_t const nd, size_t const in_size, size_t const out_size,
    hipStream_t const stream)
{
    return tomoRollaxis<double>(
        d_in, d_out, in_shape, in_shape_len, in_strides, in_strides_len,
        axis, start, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSwapaxesH(
    __half_raw const *d_in, __half_raw *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t axis1, size_t axis2,
    size_t nd, size_t in_size, size_t out_size,
    hipStream_t stream)
{
    return tomoSwapaxes<__half_raw>(
        d_in, d_out, in_shape, in_shape_len, out_shape, out_shape_len,
        in_strides, in_strides_len, out_strides, out_strides_len,
        axis1, axis2, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSwapaxesB(
    __hip_bfloat16_raw const *d_in, __hip_bfloat16_raw *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t axis1, size_t axis2,
    size_t nd, size_t in_size, size_t out_size,
    hipStream_t stream)
{
    return tomoSwapaxes<__hip_bfloat16_raw>(
        d_in, d_out, in_shape, in_shape_len, out_shape, out_shape_len,
        in_strides, in_strides_len, out_strides, out_strides_len,
        axis1, axis2, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSwapaxesF(
    float const *d_in, float *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t axis1, size_t axis2,
    size_t nd, size_t in_size, size_t out_size,
    hipStream_t stream)
{
    return tomoSwapaxes<float>(
        d_in, d_out, in_shape, in_shape_len, out_shape, out_shape_len,
        in_strides, in_strides_len, out_strides, out_strides_len,
        axis1, axis2, nd, in_size, out_size, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoSwapaxesD(
    double const *d_in, double *d_out,
    size_t const *in_shape, size_t in_shape_len,
    size_t const *out_shape, size_t out_shape_len,
    size_t const *in_strides, size_t in_strides_len,
    size_t const *out_strides, size_t out_strides_len,
    size_t axis1, size_t axis2,
    size_t nd, size_t in_size, size_t out_size,
    hipStream_t stream)
{
    return tomoSwapaxes<double>(
        d_in, d_out, in_shape, in_shape_len, out_shape, out_shape_len,
        in_strides, in_strides_len, out_strides, out_strides_len,
        axis1, axis2, nd, in_size, out_size, stream);
}

// __half version for im2col
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoIm2colH(
    __half_raw const *d_img, __half_raw *d_col,
    size_t n, size_t c, size_t h, size_t w,
    size_t kh, size_t kw,
    size_t sy, size_t sx,
    size_t ph, size_t pw,
    size_t dy, size_t dx,
    hipStream_t stream)
{
    return tomoIm2col<__half_raw>(
        d_img, d_col,
        n, c, h, w,
        kh, kw,
        sy, sx,
        ph, pw,
        dy, dx,
        stream);
}

// __hip_bfloat16 version for im2col
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoIm2colB(
    __hip_bfloat16_raw const *d_img, __hip_bfloat16_raw *d_col,
    size_t n, size_t c, size_t h, size_t w,
    size_t kh, size_t kw,
    size_t sy, size_t sx,
    size_t ph, size_t pw,
    size_t dy, size_t dx,
    hipStream_t stream)
{
    return tomoIm2col<__hip_bfloat16_raw>(
        d_img, d_col,
        n, c, h, w,
        kh, kw,
        sy, sx,
        ph, pw,
        dy, dx,
        stream);
}

// float version for im2col
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoIm2colF(
    float const *d_img, float *d_col,
    size_t n, size_t c, size_t h, size_t w,
    size_t kh, size_t kw,
    size_t sy, size_t sx,
    size_t ph, size_t pw,
    size_t dy, size_t dx,
    hipStream_t stream)
{
    return tomoIm2col<float>(
        d_img, d_col,
        n, c, h, w,
        kh, kw,
        sy, sx,
        ph, pw,
        dy, dx,
        stream);
}

// double version for im2col
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoIm2colD(
    double const *d_img, double *d_col,
    size_t n, size_t c, size_t h, size_t w,
    size_t kh, size_t kw,
    size_t sy, size_t sx,
    size_t ph, size_t pw,
    size_t dy, size_t dx,
    hipStream_t stream)
{
    return tomoIm2col<double>(
        d_img, d_col,
        n, c, h, w,
        kh, kw,
        sy, sx,
        ph, pw,
        dy, dx,
        stream);
}

// __half version for col2im
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoCol2imH(
    __half_raw const *d_col, __half_raw *d_img,
    size_t n, size_t c, size_t h, size_t w,
    size_t kh, size_t kw,
    size_t sy, size_t sx,
    size_t ph, size_t pw,
    size_t dx, size_t dy,
    hipStream_t stream)
{
    return tomoCol2im<__half_raw>(
        d_col, d_img,
        n, c, h, w,
        kh, kw,
        sy, sx,
        ph, pw,
        dx, dy,
        stream);
}

// __hip_bfloat16 version for col2im
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoCol2imB(
    __hip_bfloat16_raw const *d_col, __hip_bfloat16_raw *d_img,
    size_t n, size_t c, size_t h, size_t w,
    size_t kh, size_t kw,
    size_t sy, size_t sx,
    size_t ph, size_t pw,
    size_t dx, size_t dy,
    hipStream_t stream)
{
    return tomoCol2im<__hip_bfloat16_raw>(
        d_col, d_img,
        n, c, h, w,
        kh, kw,
        sy, sx,
        ph, pw,
        dx, dy,
        stream);
}

// float version for col2im
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoCol2imF(
    float const *d_col, float *d_img,
    size_t n, size_t c, size_t h, size_t w,
    size_t kh, size_t kw,
    size_t sy, size_t sx,
    size_t ph, size_t pw,
    size_t dx, size_t dy,
    hipStream_t stream)
{
    return tomoCol2im<float>(
        d_col, d_img,
        n, c, h, w,
        kh, kw,
        sy, sx,
        ph, pw,
        dx, dy,
        stream);
}

// double version for col2im
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoCol2imD(
    double const *d_col, double *d_img,
    size_t n, size_t c, size_t h, size_t w,
    size_t kh, size_t kw,
    size_t sy, size_t sx,
    size_t ph, size_t pw,
    size_t dx, size_t dy,
    hipStream_t stream)
{
    return tomoCol2im<double>(
        d_col, d_img,
        n, c, h, w,
        kh, kw,
        sy, sx,
        ph, pw,
        dx, dy,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoIm2col1dH(
    __half_raw const *d_in,
    __half_raw *d_col,
    size_t n, size_t c, size_t l,
    size_t k,
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    return tomoIm2col1d<__half_raw>(
        d_in, d_col,
        n, c, l, k,
        stride, pad, dilation,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoIm2col1dB(
    __hip_bfloat16_raw const *d_in,
    __hip_bfloat16_raw *d_col,
    size_t n, size_t c, size_t l,
    size_t k,
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    return tomoIm2col1d<__hip_bfloat16_raw>(
        d_in, d_col,
        n, c, l, k,
        stride, pad, dilation,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoIm2col1dF(
    float const *d_in,
    float *d_col,
    size_t n, size_t c, size_t l,
    size_t k,
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    return tomoIm2col1d<float>(
        d_in, d_col,
        n, c, l, k,
        stride, pad, dilation,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoIm2col1dD(
    double const *d_in,
    double *d_col,
    size_t n, size_t c, size_t l,
    size_t k,
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    return tomoIm2col1d<double>(
        d_in, d_col,
        n, c, l, k,
        stride, pad, dilation,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoCol2im1dH(
    __half_raw const *d_col,
    __half_raw *d_in,
    size_t n, size_t c, size_t l,
    size_t k,
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    return tomoCol2im1d<__half_raw>(
        d_col, d_in,
        n, c, l, k,
        stride, pad, dilation,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoCol2im1dB(
    __hip_bfloat16_raw const *d_col,
    __hip_bfloat16_raw *d_in,
    size_t n, size_t c, size_t l,
    size_t k,
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    return tomoCol2im1d<__hip_bfloat16_raw>(
        d_col, d_in,
        n, c, l, k,
        stride, pad, dilation,
        stream);
}
TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoCol2im1dF(
    float const *d_col,
    float *d_in,
    size_t n, size_t c, size_t l,
    size_t k,
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    return tomoCol2im1d<float>(
        d_col, d_in,
        n, c, l, k,
        stride, pad, dilation,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoCol2im1dD(
    double const *d_col,
    double *d_in,
    size_t n, size_t c, size_t l,
    size_t k,
    size_t stride,
    size_t pad,
    size_t dilation,
    hipStream_t stream)
{
    return tomoCol2im1d<double>(
        d_col, d_in,
        n, c, l, k,
        stride, pad, dilation,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoArgmaxH(
    const __half_raw *d_in,
    size_t *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t const *out_strides,
    size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoArgmax<__half_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        out_size, nd, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoArgmaxB(
    const __hip_bfloat16_raw *d_in,
    size_t *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t const *out_strides,
    size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoArgmax<__hip_bfloat16_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        out_size, nd, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoArgmaxF(
    const float *d_in,
    size_t *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t const *out_strides,
    size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoArgmax<float>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        out_size, nd, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoArgmaxD(
    const double *d_in,
    size_t *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t const *out_strides,
    size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoArgmax<double>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        out_size, nd, stream);
}

// Argmin wrappers

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoArgminH(
    const __half_raw *d_in,
    size_t *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t const *out_strides,
    size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoArgmin<__half_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        out_size, nd, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoArgminB(
    const __hip_bfloat16_raw *d_in,
    size_t *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t const *out_strides,
    size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoArgmin<__hip_bfloat16_raw>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        out_size, nd, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoArgminF(
    const float *d_in,
    size_t *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t const *out_strides,
    size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoArgmin<float>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        out_size, nd, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoArgminD(
    const double *d_in,
    size_t *d_out,
    size_t const *in_shape,
    size_t in_shape_len,
    size_t const *out_shape,
    size_t out_shape_len,
    size_t const *in_strides,
    size_t in_strides_len,
    size_t const *out_strides,
    size_t out_strides_len,
    size_t out_size,
    size_t nd,
    hipStream_t stream)
{
    return tomoArgmin<double>(
        d_in, d_out,
        in_shape, in_shape_len,
        out_shape, out_shape_len,
        in_strides, in_strides_len,
        out_strides, out_strides_len,
        out_size, nd, stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxPool2dForwardH(
    __half_raw const *d_in, __half_raw *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoMaxPool2dForward<__half_raw>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxPool2dForwardB(
    __hip_bfloat16_raw const *d_in, __hip_bfloat16_raw *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoMaxPool2dForward<__hip_bfloat16_raw>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxPool2dForwardF(
    float const *d_in, float *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoMaxPool2dForward<float>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxPool2dForwardD(
    double const *d_in, double *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoMaxPool2dForward<double>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxPool2dBackwardH(
    const __half_raw *input,
    const __half_raw *gradOut,
    __half_raw *gradIn,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW,
    hipStream_t stream)
{
    return tomoMaxPool2dBackward<__half_raw>(
        input, gradOut, gradIn,
        N, C, H, W,
        outH, outW,
        kernelH, kernelW,
        strideH, strideW,
        padH, padW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxPool2dBackwardB(
    const __hip_bfloat16_raw *input,
    const __hip_bfloat16_raw *gradOut,
    __hip_bfloat16_raw *gradIn,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW,
    hipStream_t stream)
{
    return tomoMaxPool2dBackward<__hip_bfloat16_raw>(
        input, gradOut, gradIn,
        N, C, H, W,
        outH, outW,
        kernelH, kernelW,
        strideH, strideW,
        padH, padW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxPool2dBackwardF(
    const float *input,
    const float *gradOut,
    float *gradIn,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW,
    hipStream_t stream)
{
    return tomoMaxPool2dBackward<float>(
        input, gradOut, gradIn,
        N, C, H, W,
        outH, outW,
        kernelH, kernelW,
        strideH, strideW,
        padH, padW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoMaxPool2dBackwardD(
    const double *input,
    const double *gradOut,
    double *gradIn,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kernelH, size_t kernelW,
    size_t strideH, size_t strideW,
    size_t padH, size_t padW,
    hipStream_t stream)
{
    return tomoMaxPool2dBackward<double>(
        input, gradOut, gradIn,
        N, C, H, W,
        outH, outW,
        kernelH, kernelW,
        strideH, strideW,
        padH, padW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoAvgPool2dForwardH(
    __half_raw const *d_in, __half_raw *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoAvgPool2dForward<__half_raw>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoAvgPool2dForwardB(
    __hip_bfloat16_raw const *d_in, __hip_bfloat16_raw *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoAvgPool2dForward<__hip_bfloat16_raw>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoAvgPool2dForwardF(
    float const *d_in, float *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoAvgPool2dForward<float>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoAvgPool2dForwardD(
    double const *d_in, double *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoAvgPool2dForward<double>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoAvgPool2dBackwardH(
    __half_raw const *d_in, __half_raw *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoAvgPool2dBackward<__half_raw>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoAvgPool2dBackwardB(
    __hip_bfloat16_raw const *d_in, __hip_bfloat16_raw *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoAvgPool2dBackward<__hip_bfloat16_raw>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoAvgPool2dBackwardF(
    float const *d_in, float *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoAvgPool2dBackward<float>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoAvgPool2dBackwardD(
    double const *d_in, double *d_out,
    size_t N, size_t C, size_t H, size_t W,
    size_t outH, size_t outW,
    size_t kH, size_t kW,
    size_t sH, size_t sW,
    size_t pH, size_t pW,
    hipStream_t stream)
{
    return tomoAvgPool2dBackward<double>(
        d_in, d_out,
        N, C, H, W,
        outH, outW,
        kH, kW,
        sH, sW,
        pH, pW,
        stream);
}