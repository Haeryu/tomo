#include "hip/hip_runtime.h"
#define TOMO_OPS_EXPORTS
#include "tomo_dll.h"

#include "elemwise.h"

#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include "float_op.cuh"

hipError_t tomoElemwise(auto *a,
                         auto const *b,
                         size_t len,
                         auto fn_map,
                         hipStream_t stream)
{
    using T = std::remove_cvref_t<decltype(*a)>;

    if (len == 0)
    {
        return hipErrorInvalidValue;
    }

    try
    {
        thrust::transform(thrust::cuda::par_nosync.on(stream), a, a + len, b, a, fn_map);
    }
    catch (const thrust::system_error &e)
    {
        if (e.code().category() == thrust::cuda_category())
        {
            return static_cast<hipError_t>(e.code().value());
        }
        else
        {
            return hipErrorUnknown;
        }
    }
    catch (...)
    {
        return hipErrorUnknown;
    }

    return hipSuccess;
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoProductH(__half_raw *a, __half_raw const *b, size_t len, hipStream_t stream)
{
    return tomoElemwise(a, b, len, thrust::multiplies<__half_raw>(), stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoProductB(__hip_bfloat16_raw *a, __hip_bfloat16_raw const *b, size_t len, hipStream_t stream)
{
    return tomoElemwise(a, b, len, thrust::multiplies<__hip_bfloat16_raw>(), stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoProductF(float *a, float const *b, size_t len, hipStream_t stream)
{
    return tomoElemwise(a, b, len, thrust::multiplies<float>(), stream);
}

TOMO_EXTERN_C TOMO_OPS_API hipError_t tomoProductD(double *a, double const *b, size_t len, hipStream_t stream)
{
    return tomoElemwise(a, b, len, thrust::multiplies<double>(), stream);
}